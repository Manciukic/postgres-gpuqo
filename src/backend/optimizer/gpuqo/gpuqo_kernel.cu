
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(float * a, float * b, const int N) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t j = blockIdx.y * blockDim.y + threadIdx.y;
    size_t k = blockIdx.z * blockDim.z + threadIdx.z;

    if (1 < i && 1 < j && 1 < k && i < N - 1 && j < N - 1 && k < N - 1) {
        a[i * N + j * N + k] = 0.8 * (b[(i - 1) * N + j * N + k] + b[(i + 1) * N + N * j + k] + b[i * N + (j - 1) * N + k] +
                               b[i * N + (j + 1) * N + k] + b[i * N + j * N + k-1] + b[i * N + j * N  + k + 1]);
    }
}


extern "C" void perform_stencil(float * a, float * b, const int N) {
    float * d_a;
    float * d_b;

    hipEvent_t start, stop;
    float       elapsedTime;

    /* begin timing */
    hipEventCreate(&start);
    hipEventRecord(start, 0);

    hipMalloc(&d_a, sizeof(float) * N * N * N);
    hipMalloc(&d_b, sizeof(float) * N * N * N);

    hipMemcpy(d_a, a, sizeof(float) * N * N * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N * N * N, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y, N / threadsPerBlock.z);

    kernel <<<numBlocks, threadsPerBlock>>>(d_a, d_b, N);
    hipMemcpy(a,d_a, sizeof(float) * N * N * N, hipMemcpyDeviceToHost);
    /* end timing */
    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Execution time: %f seconds\n", elapsedTime / 1000);
    hipFree(d_a);
    hipFree(d_b);
}