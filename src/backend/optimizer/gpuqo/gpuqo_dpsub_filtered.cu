#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
 *
 * gpuqo_dpsub_filtered.cu
 *      declarations necessary for dpsub_filtered_iteration
 *
 * src/backend/optimizer/gpuqo/gpuqo_dpsub_filtered.cu
 *
 *-------------------------------------------------------------------------
 */

#include <iostream>
#include <cmath>
#include <cstdint>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/tabulate.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/tuple.h>
#include <thrust/system/system_error.h>
#include <thrust/distance.h>

#include "optimizer/gpuqo_common.h"

#include "optimizer/gpuqo.cuh"
#include "optimizer/gpuqo_timing.cuh"
#include "optimizer/gpuqo_debug.cuh"
#include "optimizer/gpuqo_cost.cuh"
#include "optimizer/gpuqo_filter.cuh"
#include "optimizer/gpuqo_binomial.cuh"
#include "optimizer/gpuqo_query_tree.cuh"
#include "optimizer/gpuqo_dpsub.cuh"

// user-configured variables
bool gpuqo_dpsub_filter_enable;
int gpuqo_dpsub_filter_threshold;
int gpuqo_dpsub_filter_cpu_enum_threshold;
int gpuqo_dpsub_filter_keys_overprovisioning;

/* unrankDPSub
 *
 *	 unrank algorithm for DPsub GPU variant. 
 */
struct unrankFilteredDPSub : public thrust::unary_function< uint64_t, RelationID >
{
    thrust::device_ptr<uint64_t> binoms;
    int sq;
    int qss;
    uint64_t offset;
public:
    unrankFilteredDPSub(
        int _sq,
        thrust::device_ptr<uint64_t> _binoms,
        int _qss,
        uint64_t _offset
    ) : sq(_sq), binoms(_binoms), qss(_qss), offset(_offset)
    {}
 
    __device__
    RelationID operator()(uint64_t tid)
    {
        uint64_t sid = tid + offset;
        RelationID s = dpsub_unrank_sid(sid, qss, sq, binoms.get());
        
        LOG_DEBUG("[%llu] s=%llu\n", tid, s);
        
        RelationID relid = s<<1;
        return relid;
    }
 };
 
 /* evaluateDPSub
  *
  *	 evaluation algorithm for DPsub GPU variant with partial pruning
  */
template<typename BinaryFunction>
struct evaluateFilteredDPSub : public thrust::unary_function< uint64_t, thrust::tuple<RelationID, JoinRelation> >
{
    thrust::device_ptr<RelationID> pending_keys;
    int sq;
    int qss;
    uint64_t n_pending_sets;
    int n_pairs;
    BinaryFunction enum_functor;
public:
    evaluateFilteredDPSub(
        thrust::device_ptr<RelationID> _pending_keys,
        BinaryFunction _enum_functor,
        int _sq,
        int _qss,
        uint64_t _n_pending_sets,
        int _n_pairs
    ) : pending_keys(_pending_keys), 
        enum_functor(_enum_functor), sq(_sq), 
        qss(_qss), n_pending_sets(_n_pending_sets), n_pairs(_n_pairs)
    {}

    __device__
    thrust::tuple<RelationID, JoinRelation>  operator()(uint64_t tid)
    {
        uint64_t splits_per_qs = ceil_div((1<<qss) - 2, n_pairs);
        uint64_t rid = n_pending_sets - 1 - (tid / splits_per_qs);
        uint64_t cid = tid % splits_per_qs;
    
        RelationID relid = pending_keys[rid];

        LOG_DEBUG("[%llu] splits_per_qs=%llu, rid=%llu, cid=[%llu,%llu), relid=%llu\n", tid, splits_per_qs, rid, cid, cid+n_pairs, relid);
        
        JoinRelation jr_out = enum_functor(relid, cid);
        return thrust::make_tuple<RelationID, JoinRelation>(relid, jr_out);
    }
};

template<typename enum_functor>
int dpsub_filtered_iteration(int iter, dpsub_iter_param_t &params){   
    int n_iters = 0;
    uint64_t set_offset = 0;
    uint64_t n_pending_sets = 0;
    while (set_offset < params.n_sets){
        uint64_t n_remaining_sets = params.n_sets - set_offset;
        
        while(n_pending_sets < gpuqo_dpsub_n_parallel
                && n_remaining_sets > 0){
            uint64_t n_tab_sets;

            if (n_remaining_sets > PENDING_KEYS_SIZE-n_pending_sets){
                n_tab_sets = PENDING_KEYS_SIZE-n_pending_sets;
            } else {
                n_tab_sets = n_remaining_sets;
            }

            if (n_tab_sets == 1){
                // if it's only one it's the last one so it's valid
                params.gpu_pending_keys[n_pending_sets] = params.out_relid;
                n_pending_sets += 1;
            } else if (n_tab_sets <= gpuqo_dpsub_filter_cpu_enum_threshold) {
                // fill (valid) pending keys on CPU
                // if they are too few do not bother going to GPU

                START_TIMING(unrank);
                thrust::host_vector<RelationID> relids(n_tab_sets);
                uint64_t n_valid_relids = 0;
                for (uint64_t sid=0; sid < n_tab_sets; sid++){
                    RelationID relid = dpsub_unrank_sid(sid, iter, params.n_rels, params.binoms.data()) << 1;
                    if (is_connected(relid, params.base_rels, params.n_rels, params.edge_table)){
                        relids[n_valid_relids++] = relid; 
                    }
                }
                thrust::copy(relids.begin(), relids.begin()+n_valid_relids, params.gpu_pending_keys.begin()+n_pending_sets);

                n_pending_sets += n_valid_relids;
                STOP_TIMING(unrank);
            } else {
                // fill pending keys and filter on GPU 
                START_TIMING(unrank);
                thrust::tabulate(
                    params.gpu_pending_keys.begin()+n_pending_sets,
                    params.gpu_pending_keys.begin()+(n_pending_sets+n_tab_sets),
                    unrankFilteredDPSub(
                        params.n_rels,
                        params.gpu_binoms.data(),
                        iter,
                        set_offset
                    ) 
                );
                STOP_TIMING(unrank);

                START_TIMING(filter);
                auto keys_end_iter = thrust::remove_if(
                    params.gpu_pending_keys.begin()+n_pending_sets,
                    params.gpu_pending_keys.begin()+(n_pending_sets+n_tab_sets),
                    filterDisconnectedRelations(
                        params.gpu_base_rels.data(), 
                        params.n_rels,
                        params.gpu_edge_table.data()
                    )
                );
                STOP_TIMING(filter);

                n_pending_sets = thrust::distance(
                    params.gpu_pending_keys.begin(),
                    keys_end_iter
                );
            } 

            set_offset += n_tab_sets;
            n_remaining_sets -= n_tab_sets;
        }   

        uint64_t n_joins_per_thread;
        uint64_t n_sets_per_iteration;
        uint64_t factor = gpuqo_dpsub_n_parallel / n_pending_sets;
        if (factor < 1){ // n_sets > gpuqo_dpsub_n_parallel
            n_joins_per_thread = params.n_joins_per_set;
            n_sets_per_iteration = gpuqo_dpsub_n_parallel;
        } else{
            n_sets_per_iteration = n_pending_sets;
            n_joins_per_thread = ceil_div(params.n_joins_per_set, factor);
        }     
        uint64_t threads_per_set = ceil_div(params.n_joins_per_set, n_joins_per_thread);   

        LOG_PROFILE("n_joins_per_thread=%llu, n_sets_per_iteration=%llu, threads_per_set=%llu\n",
            n_joins_per_thread,
            n_sets_per_iteration,
            threads_per_set
        );

        // do not empty all pending sets if there are some sets still to 
        // evaluate, since I will do them in the next iteration
        // If no sets remain, then I will empty all pending
        while (n_pending_sets >= gpuqo_dpsub_n_parallel 
            || (n_pending_sets > 0 && n_remaining_sets == 0)
        ){
            uint64_t n_eval_sets = min(n_sets_per_iteration, n_pending_sets);
            uint64_t n_threads = n_eval_sets * threads_per_set;
            
            START_TIMING(compute);
            thrust::tabulate(
                thrust::make_zip_iterator(thrust::make_tuple(
                    params.gpu_scratchpad_keys.begin(),
                    params.gpu_scratchpad_vals.begin()
                )),
                thrust::make_zip_iterator(thrust::make_tuple(
                    params.gpu_scratchpad_keys.begin()+n_threads,
                    params.gpu_scratchpad_vals.begin()+n_threads
                )),
                evaluateFilteredDPSub<enum_functor>(
                    params.gpu_pending_keys.data(),
                    enum_functor(
                        params.gpu_memo_vals.data(),
                        params.gpu_base_rels.data(),
                        params.n_rels,
                        params.gpu_edge_table.data(),
                        n_joins_per_thread
                    ),
                    params.n_rels,
                    iter,
                    n_pending_sets,
                    n_joins_per_thread
                ) 
            );
            STOP_TIMING(compute);

            LOG_DEBUG("After tabulate\n");
            DUMP_VECTOR(params.gpu_scratchpad_keys.begin(), params.gpu_scratchpad_keys.begin()+n_threads);
            DUMP_VECTOR(params.gpu_scratchpad_vals.begin(), params.gpu_scratchpad_vals.begin()+n_threads);

            dpsub_prune_scatter(n_joins_per_thread, n_threads, params);

            n_pending_sets -= n_eval_sets;
        }

        n_iters++;
    }

    return n_iters;
}

template int dpsub_filtered_iteration<dpsubEnumerateAllSubs>(int iter, dpsub_iter_param_t &params);
template int dpsub_filtered_iteration<dpsubEnumerateCsg>(int iter, dpsub_iter_param_t &params);
