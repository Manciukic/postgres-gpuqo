#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
 *
 * gpuqo_dpsub_filtered.cu
 *      declarations necessary for dpsub_filtered_iteration
 *
 * src/backend/optimizer/gpuqo/gpuqo_dpsub_filtered.cu
 *
 *-------------------------------------------------------------------------
 */

#include <iostream>
#include <cmath>
#include <cstdint>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/tabulate.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/tuple.h>
#include <thrust/system/system_error.h>
#include <thrust/distance.h>

#include "optimizer/gpuqo_common.h"

#include "gpuqo.cuh"
#include "gpuqo_timing.cuh"
#include "gpuqo_debug.cuh"
#include "gpuqo_cost.cuh"
#include "gpuqo_filter.cuh"
#include "gpuqo_binomial.cuh"
#include "gpuqo_query_tree.cuh"
#include "gpuqo_dpsub.cuh"
#include "gpuqo_dpsub_enum_all_subs.cuh"
#include "gpuqo_dpsub_csg.cuh"

// user-configured variables
bool gpuqo_dpsub_filter_enable;
int gpuqo_dpsub_filter_threshold;
int gpuqo_dpsub_filter_cpu_enum_threshold;
int gpuqo_dpsub_filter_keys_overprovisioning;

/* unrankDPSub
 *
 *	 unrank algorithm for DPsub GPU variant. 
 */
struct unrankFilteredDPSub : public thrust::unary_function< uint64_t, RelationID >
{
    thrust::device_ptr<uint64_t> binoms;
    int sq;
    int qss;
    uint64_t offset;
public:
    unrankFilteredDPSub(
        int _sq,
        thrust::device_ptr<uint64_t> _binoms,
        int _qss,
        uint64_t _offset
    ) : sq(_sq), binoms(_binoms), qss(_qss), offset(_offset)
    {}
 
    __device__
    RelationID operator()(uint64_t tid)
    {
        uint64_t sid = tid + offset;
        RelationID s = dpsub_unrank_sid(sid, qss, sq, binoms.get());
        
        LOG_DEBUG("[%llu] s=%llu\n", tid, s);
        
        RelationID relid = s<<1;
        return relid;
    }
 };
 
 /* evaluateDPSub
  *
  *	 evaluation algorithm for DPsub GPU variant with partial pruning
  */
template<typename BinaryFunction>
struct evaluateFilteredDPSub : public thrust::unary_function< uint64_t, thrust::tuple<RelationID, JoinRelation> >
{
    thrust::device_ptr<RelationID> pending_keys;
    int sq;
    int qss;
    uint64_t n_pending_sets;
    int n_splits;
    BinaryFunction enum_functor;
public:
    evaluateFilteredDPSub(
        thrust::device_ptr<RelationID> _pending_keys,
        BinaryFunction _enum_functor,
        int _sq,
        int _qss,
        uint64_t _n_pending_sets,
        int _n_splits
    ) : pending_keys(_pending_keys), 
        enum_functor(_enum_functor), sq(_sq), 
        qss(_qss), n_pending_sets(_n_pending_sets), n_splits(_n_splits)
    {}

    __device__
    thrust::tuple<RelationID, JoinRelation>  operator()(uint64_t tid)
    {
        uint64_t rid = n_pending_sets - 1 - (tid / n_splits);
        uint64_t cid = tid % n_splits;
    
        RelationID relid = pending_keys[rid];

        LOG_DEBUG("[%llu] n_splits=%d, rid=%llu, cid=%llu, relid=%llu\n", 
                tid, n_splits, rid, cid, relid);
        
        JoinRelation jr_out = enum_functor(relid, cid);
        Assert(jr_out.id == BMS64_EMPTY || jr_out.id == relid);
        return thrust::make_tuple<RelationID, JoinRelation>(relid, jr_out);
    }
};

int dpsub_filtered_iteration(int iter, dpsub_iter_param_t &params){   
    int n_iters = 0;
    uint64_t set_offset = 0;
    uint64_t n_pending_sets = 0;
    while (set_offset < params.n_sets){
        uint64_t n_remaining_sets = params.n_sets - set_offset;
        
        while(n_pending_sets < gpuqo_dpsub_n_parallel
                && n_remaining_sets > 0){
            uint64_t n_tab_sets;

            if (n_remaining_sets > PENDING_KEYS_SIZE-n_pending_sets){
                n_tab_sets = PENDING_KEYS_SIZE-n_pending_sets;
            } else {
                n_tab_sets = n_remaining_sets;
            }

            if (n_tab_sets == 1){
                // if it's only one it's the last one so it's valid
                params.gpu_pending_keys[n_pending_sets] = params.out_relid;
                n_pending_sets += 1;
            } else if (n_tab_sets <= gpuqo_dpsub_filter_cpu_enum_threshold) {
                // fill (valid) pending keys on CPU
                // if they are too few do not bother going to GPU

                START_TIMING(unrank);
                thrust::host_vector<RelationID> relids(n_tab_sets);
                uint64_t n_valid_relids = 0;
                for (uint64_t sid=0; sid < n_tab_sets; sid++){
                    RelationID relid = dpsub_unrank_sid(sid, iter, params.info->n_rels, params.binoms.data()) << 1;
                    if (is_connected(relid, params.info)){
                        relids[n_valid_relids++] = relid; 
                    }
                }
                thrust::copy(relids.begin(), relids.begin()+n_valid_relids, params.gpu_pending_keys.begin()+n_pending_sets);

                n_pending_sets += n_valid_relids;
                STOP_TIMING(unrank);
            } else {
                // fill pending keys and filter on GPU 
                START_TIMING(unrank);
                thrust::tabulate(
                    params.gpu_pending_keys.begin()+n_pending_sets,
                    params.gpu_pending_keys.begin()+(n_pending_sets+n_tab_sets),
                    unrankFilteredDPSub(
                        params.info->n_rels,
                        params.gpu_binoms.data(),
                        iter,
                        set_offset
                    ) 
                );
                STOP_TIMING(unrank);

                START_TIMING(filter);
                auto keys_end_iter = thrust::remove_if(
                    params.gpu_pending_keys.begin()+n_pending_sets,
                    params.gpu_pending_keys.begin()+(n_pending_sets+n_tab_sets),
                    filterDisconnectedRelations(params.info)
                );
                STOP_TIMING(filter);

                n_pending_sets = thrust::distance(
                    params.gpu_pending_keys.begin(),
                    keys_end_iter
                );
            } 

            set_offset += n_tab_sets;
            n_remaining_sets -= n_tab_sets;
        }   

        uint64_t n_joins_per_thread;
        uint64_t n_sets_per_iteration;
        uint64_t threads_per_set;
        uint64_t factor = gpuqo_dpsub_n_parallel / n_pending_sets;

        if (factor < 32 || params.n_joins_per_set <= 32){
            threads_per_set = 32;
        } else{
            threads_per_set = BMS64_HIGHEST(min(factor, params.n_joins_per_set));
        }
        
        n_joins_per_thread = ceil_div(params.n_joins_per_set, threads_per_set);
        n_sets_per_iteration = min(gpuqo_dpsub_n_parallel / threads_per_set, n_pending_sets);

        LOG_PROFILE("n_joins_per_thread=%llu, n_sets_per_iteration=%llu, threads_per_set=%llu, factor=%llu\n",
            n_joins_per_thread,
            n_sets_per_iteration,
            threads_per_set,
            factor
        );

        bool use_csg = (gpuqo_dpsub_csg_enable && n_joins_per_thread >= gpuqo_dpsub_csg_threshold);

        if (use_csg){
            LOG_PROFILE("Using CSG enumeration\n");
        } else{
            LOG_PROFILE("Using all subsets enumeration\n");
        }

        // do not empty all pending sets if there are some sets still to 
        // evaluate, since I will do them in the next iteration
        // If no sets remain, then I will empty all pending
        while (n_pending_sets >= gpuqo_dpsub_n_parallel 
            || (n_pending_sets > 0 && n_remaining_sets == 0)
        ){
            uint64_t n_eval_sets = min(n_sets_per_iteration, n_pending_sets);
            uint64_t n_threads = n_eval_sets * threads_per_set;

            START_TIMING(compute);
            if (use_csg) {
                thrust::tabulate(
                    thrust::make_zip_iterator(thrust::make_tuple(
                        params.gpu_scratchpad_keys.begin(),
                        params.gpu_scratchpad_vals.begin()
                    )),
                    thrust::make_zip_iterator(thrust::make_tuple(
                        params.gpu_scratchpad_keys.begin()+n_threads,
                        params.gpu_scratchpad_vals.begin()+n_threads
                    )),
                    evaluateFilteredDPSub<dpsubEnumerateCsg>(
                        params.gpu_pending_keys.data(),
                        dpsubEnumerateCsg(
                            params.gpu_memo_vals.data(),
                            params.info,
                            threads_per_set
                        ),
                        params.info->n_rels,
                        iter,
                        n_pending_sets,
                        threads_per_set
                    )                
                );
            } else {
                thrust::tabulate(
                    thrust::make_zip_iterator(thrust::make_tuple(
                        params.gpu_scratchpad_keys.begin(),
                        params.gpu_scratchpad_vals.begin()
                    )),
                    thrust::make_zip_iterator(thrust::make_tuple(
                        params.gpu_scratchpad_keys.begin()+n_threads,
                        params.gpu_scratchpad_vals.begin()+n_threads
                    )),
                    evaluateFilteredDPSub<dpsubEnumerateAllSubs>(
                        params.gpu_pending_keys.data(),
                        dpsubEnumerateAllSubs(
                            params.gpu_memo_vals.data(),
                            params.info,
                            threads_per_set
                        ),
                        params.info->n_rels,
                        iter,
                        n_pending_sets,
                        threads_per_set
                    )             
                );
            }            
            STOP_TIMING(compute);

            LOG_DEBUG("After tabulate\n");
            DUMP_VECTOR(params.gpu_scratchpad_keys.begin(), params.gpu_scratchpad_keys.begin()+n_threads);
            DUMP_VECTOR(params.gpu_scratchpad_vals.begin(), params.gpu_scratchpad_vals.begin()+n_threads);

            dpsub_prune_scatter(threads_per_set, n_threads, params);

            n_pending_sets -= n_eval_sets;
        }

        n_iters++;
    }

    return n_iters;
}
