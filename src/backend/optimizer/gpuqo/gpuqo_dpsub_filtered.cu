#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
 *
 * gpuqo_dpsub_filtered.cu
 *      declarations necessary for dpsub_filtered_iteration
 *
 * src/backend/optimizer/gpuqo/gpuqo_dpsub_filtered.cu
 *
 *-------------------------------------------------------------------------
 */

#include <iostream>
#include <cmath>
#include <cstdint>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/tabulate.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/tuple.h>
#include <thrust/system/system_error.h>
#include <thrust/distance.h>

#include "optimizer/gpuqo_common.h"

#include "gpuqo.cuh"
#include "gpuqo_timing.cuh"
#include "gpuqo_debug.cuh"
#include "gpuqo_cost.cuh"
#include "gpuqo_filter.cuh"
#include "gpuqo_binomial.cuh"
#include "gpuqo_query_tree.cuh"
#include "gpuqo_dpsub.cuh"
#include "gpuqo_dpsub_enum_all_subs.cuh"
#include "gpuqo_dpsub_csg.cuh"
#include "gpuqo_dpsub_tree.cuh"

// user-configured variables
bool gpuqo_dpsub_filter_enable;
int gpuqo_dpsub_filter_threshold;
int gpuqo_dpsub_filter_cpu_enum_threshold;
int gpuqo_dpsub_filter_keys_overprovisioning;

/* unrankDPSub
 *
 *	 unrank algorithm for DPsub GPU variant. 
 */
struct unrankFilteredDPSub : public thrust::unary_function< uint32_t, RelationID >
{
    thrust::device_ptr<uint32_t> binoms;
    int sq;
    int qss;
    uint32_t offset;
public:
    unrankFilteredDPSub(
        int _sq,
        thrust::device_ptr<uint32_t> _binoms,
        int _qss,
        uint32_t _offset
    ) : sq(_sq), binoms(_binoms), qss(_qss), offset(_offset)
    {}
 
    __device__
    RelationID operator()(uint32_t tid)
    {
        uint32_t sid = tid + offset;

        // why not use shared memory?
        // I tried but improvements are small
        RelationID s = dpsub_unrank_sid(sid, qss, sq, binoms.get());
        
        LOG_DEBUG("[%u] s=%u\n", tid, s);
        
        RelationID relid = s<<1;
        return relid;
    }
 };
 
 /* evaluateDPSub
  *
  *	 evaluation algorithm for DPsub GPU variant with partial pruning
  */
template<typename BinaryFunction>
struct evaluateFilteredDPSub : public thrust::unary_function< uint32_t, thrust::tuple<RelationID, JoinRelation> >
{
    thrust::device_ptr<RelationID> pending_keys;
    int sq;
    int qss;
    uint32_t n_pending_sets;
    int n_splits;
    BinaryFunction enum_functor;
public:
    evaluateFilteredDPSub(
        thrust::device_ptr<RelationID> _pending_keys,
        BinaryFunction _enum_functor,
        int _sq,
        int _qss,
        uint32_t _n_pending_sets,
        int _n_splits
    ) : pending_keys(_pending_keys), 
        enum_functor(_enum_functor), sq(_sq), 
        qss(_qss), n_pending_sets(_n_pending_sets), n_splits(_n_splits)
    {}

    __device__
    thrust::tuple<RelationID, JoinRelation>  operator()(uint32_t tid)
    {
        uint32_t rid = n_pending_sets - 1 - (tid / n_splits);
        uint32_t cid = tid % n_splits;

        Assert(n_pending_sets-1 <= 0xFFFFFFFF - tid / n_splits);
    
        RelationID relid = pending_keys[rid];

        LOG_DEBUG("[%u] n_splits=%d, rid=%u, cid=%u, relid=%u\n", 
                tid, n_splits, rid, cid, relid);
        
        JoinRelation jr_out = enum_functor(relid, cid);
        Assert(jr_out.id == BMS32_EMPTY || jr_out.id == relid);
        return thrust::make_tuple<RelationID, JoinRelation>(relid, jr_out);
    }
};


uint32_t dpsub_generic_graph_evaluation(int iter, uint32_t n_remaining_sets,
                                    uint32_t offset, uint32_t n_pending_sets, 
                                    dpsub_iter_param_t &params)
{
    uint32_t n_joins_per_thread;
    uint32_t n_sets_per_iteration;
    uint32_t threads_per_set;
    uint32_t factor = gpuqo_n_parallel / n_pending_sets;

    if (factor < 32 || params.n_joins_per_set <= 32){
        threads_per_set = 32;
    } else{
        threads_per_set = BMS32_HIGHEST(min(factor, params.n_joins_per_set));
    }
    
    n_joins_per_thread = ceil_div(params.n_joins_per_set, threads_per_set);
    n_sets_per_iteration = min(params.scratchpad_size / threads_per_set, n_pending_sets);

    LOG_PROFILE("n_joins_per_thread=%u, n_sets_per_iteration=%u, threads_per_set=%u, factor=%u\n",
        n_joins_per_thread,
        n_sets_per_iteration,
        threads_per_set,
        factor
    );

    bool use_csg = (gpuqo_dpsub_csg_enable && n_joins_per_thread >= gpuqo_dpsub_csg_threshold);

    if (use_csg){
        LOG_PROFILE("Using CSG enumeration\n");
    } else{
        LOG_PROFILE("Using all subsets enumeration\n");
    }

    // do not empty all pending sets if there are some sets still to 
    // evaluate, since I will do them in the next iteration
    // If no sets remain, then I will empty all pending
    while (n_pending_sets >= gpuqo_n_parallel 
        || (n_pending_sets > 0 && n_remaining_sets == 0)
    ){
        uint32_t n_eval_sets = min(n_sets_per_iteration, n_pending_sets);
        uint32_t n_threads = n_eval_sets * threads_per_set;

        START_TIMING(compute);
        if (use_csg) {
            thrust::tabulate(
                thrust::make_zip_iterator(thrust::make_tuple(
                    params.gpu_scratchpad_keys.begin(),
                    params.gpu_scratchpad_vals.begin()
                )),
                thrust::make_zip_iterator(thrust::make_tuple(
                    params.gpu_scratchpad_keys.begin()+n_threads,
                    params.gpu_scratchpad_vals.begin()+n_threads
                )),
                evaluateFilteredDPSub<dpsubEnumerateCsg>(
                    params.gpu_pending_keys.data()+offset,
                    dpsubEnumerateCsg(
                        *params.memo,
                        params.info,
                        threads_per_set
                    ),
                    params.info->n_rels,
                    iter,
                    n_pending_sets,
                    threads_per_set
                )                
            );
        } else {
            thrust::tabulate(
                thrust::make_zip_iterator(thrust::make_tuple(
                    params.gpu_scratchpad_keys.begin(),
                    params.gpu_scratchpad_vals.begin()
                )),
                thrust::make_zip_iterator(thrust::make_tuple(
                    params.gpu_scratchpad_keys.begin()+n_threads,
                    params.gpu_scratchpad_vals.begin()+n_threads
                )),
                evaluateFilteredDPSub<dpsubEnumerateAllSubs>(
                    params.gpu_pending_keys.data()+offset,
                    dpsubEnumerateAllSubs(
                        *params.memo,
                        params.info,
                        threads_per_set
                    ),
                    params.info->n_rels,
                    iter,
                    n_pending_sets,
                    threads_per_set
                )             
            );
        }           
        STOP_TIMING(compute);

        LOG_DEBUG("After tabulate\n");
        DUMP_VECTOR(params.gpu_scratchpad_keys.begin(), params.gpu_scratchpad_keys.begin()+n_threads);
        DUMP_VECTOR(params.gpu_scratchpad_vals.begin(), params.gpu_scratchpad_vals.begin()+n_threads);

        dpsub_prune_scatter(threads_per_set, n_threads, params);

        n_pending_sets -= n_eval_sets;
    }

    return n_pending_sets;
}


uint32_t dpsub_tree_evaluation(int iter, uint32_t n_remaining_sets, 
                           uint32_t offset, uint32_t n_pending_sets, 
                           dpsub_iter_param_t &params)
{
    uint32_t n_joins_per_thread;
    uint32_t n_sets_per_iteration;
    uint32_t threads_per_set;
    uint32_t factor = gpuqo_n_parallel / n_pending_sets;
    uint32_t n_joins_per_set = iter; 

    threads_per_set = min(max(1, factor), n_joins_per_set);
    
    n_joins_per_thread = ceil_div(n_joins_per_set, threads_per_set);
    n_sets_per_iteration = min(params.scratchpad_size / threads_per_set, n_pending_sets);

    LOG_PROFILE("n_joins_per_thread=%u, n_sets_per_iteration=%u, threads_per_set=%u, factor=%u\n",
        n_joins_per_thread,
        n_sets_per_iteration,
        threads_per_set,
        factor
    );

    LOG_PROFILE("Using tree enumeration\n");

    // do not empty all pending sets if there are some sets still to 
    // evaluate, since I will do them in the next iteration
    // If no sets remain, then I will empty all pending
    while (n_pending_sets >= gpuqo_n_parallel 
        || (n_pending_sets > 0 && n_remaining_sets == 0)
    ){
        uint32_t n_eval_sets = min(n_sets_per_iteration, n_pending_sets);
        uint32_t n_threads = n_eval_sets * threads_per_set;

        START_TIMING(compute);
        thrust::tabulate(
            thrust::make_zip_iterator(thrust::make_tuple(
                params.gpu_scratchpad_keys.begin(),
                params.gpu_scratchpad_vals.begin()
            )),
            thrust::make_zip_iterator(thrust::make_tuple(
                params.gpu_scratchpad_keys.begin()+n_threads,
                params.gpu_scratchpad_vals.begin()+n_threads
            )),
            evaluateFilteredDPSub<dpsubEnumerateTreeSimple>(
                params.gpu_pending_keys.data()+offset,
                dpsubEnumerateTreeSimple(
                    *params.memo,
                    params.info,
                    threads_per_set
                ),
                params.info->n_rels,
                iter,
                n_pending_sets,
                threads_per_set
            )             
        );
                    
        STOP_TIMING(compute);

        LOG_PROFILE("After tabulate\n");
        DUMP_VECTOR(params.gpu_scratchpad_keys.begin(), params.gpu_scratchpad_keys.begin()+n_threads);
        DUMP_VECTOR(params.gpu_scratchpad_vals.begin(), params.gpu_scratchpad_vals.begin()+n_threads);

        dpsub_prune_scatter(threads_per_set, n_threads, params);

        n_pending_sets -= n_eval_sets;
    }

    return n_pending_sets;
}


int dpsub_filtered_iteration(int iter, dpsub_iter_param_t &params){   
    int n_iters = 0;
    uint32_t set_offset = 0;
    uint32_t n_pending_sets = 0;
    while (set_offset < params.n_sets){
        uint32_t n_remaining_sets = params.n_sets - set_offset;
        
        while(n_pending_sets < params.scratchpad_size
                && n_remaining_sets > 0){
            uint32_t n_tab_sets;

            if (n_remaining_sets > PENDING_KEYS_SIZE(params)-n_pending_sets){
                n_tab_sets = PENDING_KEYS_SIZE(params)-n_pending_sets;
            } else {
                n_tab_sets = n_remaining_sets;
            }

            if (n_tab_sets == 1){
                // if it's only one it's the last one so it's valid
                params.gpu_pending_keys[n_pending_sets] = params.out_relid;
                n_pending_sets += 1;
            } else if (n_tab_sets <= gpuqo_dpsub_filter_cpu_enum_threshold) {
                // fill (valid) pending keys on CPU
                // if they are too few do not bother going to GPU

                START_TIMING(unrank);
                thrust::host_vector<RelationID> relids(n_tab_sets);
                uint32_t n_valid_relids = 0;
                for (uint32_t sid=0; sid < n_tab_sets; sid++){
                    RelationID relid = dpsub_unrank_sid(sid, iter, params.info->n_rels, params.binoms.data()) << 1;
                    if (is_connected(relid, params.info->edge_table)){
                        relids[n_valid_relids++] = relid; 
                    }
                }
                thrust::copy(relids.begin(), relids.begin()+n_valid_relids, params.gpu_pending_keys.begin()+n_pending_sets);

                n_pending_sets += n_valid_relids;
                STOP_TIMING(unrank);
            } else {
                // fill pending keys and filter on GPU 
                START_TIMING(unrank);
                thrust::tabulate(
                    params.gpu_pending_keys.begin()+n_pending_sets,
                    params.gpu_pending_keys.begin()+(n_pending_sets+n_tab_sets),
                    unrankFilteredDPSub(
                        params.info->n_rels,
                        params.gpu_binoms.data(),
                        iter,
                        set_offset
                    ) 
                );
                STOP_TIMING(unrank);

                START_TIMING(filter);
                auto keys_end_iter = thrust::remove_if(
                    params.gpu_pending_keys.begin()+n_pending_sets,
                    params.gpu_pending_keys.begin()+(n_pending_sets+n_tab_sets),
                    filterDisconnectedRelations(params.info)
                );
                STOP_TIMING(filter);

                n_pending_sets = thrust::distance(
                    params.gpu_pending_keys.begin(),
                    keys_end_iter
                );
            } 

            set_offset += n_tab_sets;
            n_remaining_sets -= n_tab_sets;
        }  
        
        if (gpuqo_dpsub_tree_enable){
            auto middle = thrust::partition(
                params.gpu_pending_keys.begin(),
                params.gpu_pending_keys.begin()+n_pending_sets,
                findCycleInRelation(params.info)
            );

            int n_cyclic = thrust::distance(
                params.gpu_pending_keys.begin(),
                middle
            );

            LOG_PROFILE("Cyclic: %d, Trees: %d, Tot: %d\n", 
                n_cyclic, 
                n_pending_sets - n_cyclic, 
                n_pending_sets
            );

            uint32_t graph_pending = 0;
            uint32_t tree_pending = 0;

            // TODO: maybe I can run both kernels in parallel if I have few
            //       relations
            if (n_cyclic > 0){
                graph_pending = dpsub_generic_graph_evaluation(
                                    iter, n_remaining_sets, 
                                               0, n_cyclic, params);
            }

            if (n_pending_sets - n_cyclic > 0){
                tree_pending = dpsub_tree_evaluation(iter, n_remaining_sets,
                                      n_cyclic, n_pending_sets-n_cyclic, 
                                      params);
            }

            // recompact
            if (n_cyclic > 0 && tree_pending != 0){
                thrust::copy(middle, middle + tree_pending, 
                            params.gpu_pending_keys.begin() + graph_pending
                );
            }

            n_pending_sets = graph_pending + tree_pending;


        } else {
            n_pending_sets = dpsub_generic_graph_evaluation(
                                        iter, n_remaining_sets, 
                                           0, n_pending_sets, params);
        }
        
        n_iters++;
    }

    return n_iters;
}
