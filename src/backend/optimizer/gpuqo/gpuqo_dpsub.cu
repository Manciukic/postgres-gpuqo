#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
 *
 * gpuqo_dpsub.cu
 *
 * src/backend/optimizer/gpuqo/gpuqo_dpsub.cu
 *
 *-------------------------------------------------------------------------
 */

#include <iostream>
#include <cmath>
#include <cstdint>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/tabulate.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/tuple.h>
#include <thrust/system/system_error.h>
#include <thrust/distance.h>

#include "optimizer/gpuqo_common.h"

#include "optimizer/gpuqo.cuh"
#include "optimizer/gpuqo_timing.cuh"
#include "optimizer/gpuqo_debug.cuh"
#include "optimizer/gpuqo_cost.cuh"
#include "optimizer/gpuqo_filter.cuh"
#include "optimizer/gpuqo_binomial.cuh"
#include "optimizer/gpuqo_query_tree.cuh"
#include "optimizer/gpuqo_dpsub.cuh"

// relsize depends on algorithm
#define RELSIZE (sizeof(JoinRelation))

// User-configured option
int gpuqo_dpsub_n_parallel;

__device__
RelationID dpsub_unrank_sid(uint64_t sid, uint64_t qss, uint64_t sq, uint64_t* binoms){
    RelationID s = BMS64_EMPTY;
    int t = 0;
    int qss_tmp = qss, sq_tmp = sq;

    while (sq_tmp > 0 && qss_tmp > 0){
        uint64_t o = BINOM(binoms, sq, sq_tmp-1, qss_tmp-1);
        if (sid < o){
            s = BMS64_UNION(s, BMS64_NTH(t));
            qss_tmp--;
        } else {
            sid -= o;
        }
        t++;
        sq_tmp--;
    }

    return s;
}

__device__
void try_join(RelationID relid, JoinRelation &jr_out, 
            RelationID l, RelationID r, JoinRelation* memo_vals,
            BaseRelation* base_rels, int n_rels, EdgeInfo* edge_table) {
    if (l == BMS64_EMPTY || r == BMS64_EMPTY){
        return;
    }

#ifdef GPUQO_DEBUG
    printf("try_join(%llu, %llu, %llu)\n", relid, l, r);
#endif

    JoinRelation jr;
    jr.id = relid;
    jr.left_relation_id = l;
    jr.left_relation_idx = l;
    jr.right_relation_id = r;
    jr.right_relation_idx = r;
    
    JoinRelation left_rel = memo_vals[jr.left_relation_idx];
    JoinRelation right_rel = memo_vals[jr.right_relation_idx];

    // make sure those subsets were valid in a previous iteration
    if (left_rel.id == l && right_rel.id == r){
        jr.edges = BMS64_UNION(left_rel.edges, right_rel.edges);
        
        if (are_connected(left_rel, right_rel, base_rels, n_rels, edge_table)){

#ifdef GPUQO_DEBUG 
        printf("[%llu] Joining %llu and %llu\n", relid, l, r);
#endif

            jr.rows = estimate_join_rows(jr, left_rel, right_rel,
                                base_rels, n_rels, edge_table);

            jr.cost = compute_join_cost(jr, left_rel, right_rel,
                                base_rels, n_rels, edge_table);

            if (jr.cost < jr_out.cost){
                jr_out = jr;
            }
        } else {
#ifdef GPUQO_DEBUG 
            printf("[%llu] Cannot join %llu and %llu\n", relid, l, r);
#endif
        }
    } else {
#ifdef GPUQO_DEBUG 
        printf("[%llu] Invalid subsets %llu and %llu\n", relid, l, r);
#endif
    }
}


__device__
thrust::tuple<RelationID, JoinRelation> unrankEvaluateDPSub::operator()(uint64_t tid) 
{
    uint64_t splits_per_qs = ceil_div((1<<qss) - 2, n_pairs);
    uint64_t real_id = tid + offset;
    uint64_t sid = real_id / splits_per_qs;
    uint64_t cid = (real_id % splits_per_qs)*n_pairs+1;

#ifdef GPUQO_DEBUG 
    printf("[%llu] splits_per_qs=%llu, sid=%llu, cid=[%llu,%llu)\n", tid, splits_per_qs, sid, cid, cid+n_pairs);
#endif

    RelationID s = dpsub_unrank_sid(sid, qss, sq, binoms.get());

#ifdef GPUQO_DEBUG 
    printf("[%llu] s=%llu\n", tid, s);
#endif
    
    JoinRelation jr_out;
    jr_out.id = BMS64_EMPTY;
    jr_out.cost = INFD;
    RelationID relid = s<<1;
    RelationID l = BMS64_EXPAND_TO_MASK(cid, relid);
    RelationID r;

    for (int i = 0; i < n_pairs; i++){
        r = BMS64_DIFFERENCE(relid, l);
        
        try_join(relid, jr_out, l, r, 
                memo_vals.get(), base_rels.get(), sq, edge_table.get());

        l = BMS64_NEXT_SUBSET(l, relid);
    }

    return thrust::tuple<RelationID, JoinRelation>(relid, jr_out);
}

/* gpuqo_dpsub
 *
 *	 GPU query optimization using the DP size variant.
 */
extern "C"
QueryTree*
gpuqo_dpsub(BaseRelation base_rels[], int n_rels, EdgeInfo edge_table[])
{
    DECLARE_TIMING(gpuqo_dpsub);
    DECLARE_NV_TIMING(init);
    DECLARE_NV_TIMING(execute);
    
    START_TIMING(gpuqo_dpsub);
    START_TIMING(init);

    uint64_t max_memo_size = gpuqo_dpsize_max_memo_size_mb * MB / RELSIZE;
    uint64_t req_memo_size = 1ULL<<(n_rels+1);
    if (max_memo_size < req_memo_size){
        printf("Insufficient memo size\n");
        return NULL;
    }

    uint64_t memo_size = std::min(req_memo_size, max_memo_size);
    
    thrust::device_vector<BaseRelation> gpu_base_rels(base_rels, base_rels + n_rels);
    thrust::device_vector<EdgeInfo> gpu_edge_table(edge_table, edge_table + n_rels*n_rels);
    thrust::device_vector<JoinRelation> gpu_memo_vals(memo_size);
    QueryTree* out = NULL;
    RelationID out_relid = BMS64_EMPTY;

    for(int i=0; i<n_rels; i++){
        JoinRelation t;
        t.id = base_rels[i].id;
        t.left_relation_idx = 0; 
        t.left_relation_id = 0; 
        t.right_relation_idx = 0; 
        t.right_relation_id = 0; 
        t.cost = baserel_cost(base_rels[i]); 
        t.rows = base_rels[i].rows; 
        t.edges = base_rels[i].edges;
        gpu_memo_vals[base_rels[i].id] = t;

        out_relid = BMS64_UNION(out_relid, base_rels[i].id);
    }

    int binoms_size = (n_rels+1)*(n_rels+1);
    thrust::host_vector<uint64_t> binoms(binoms_size);
    precompute_binoms(binoms, n_rels);
    thrust::device_vector<uint64_t> gpu_binoms = binoms;

    // scratchpad size is increased on demand, starting from a minimum capacity
    uninit_device_vector_relid gpu_scratchpad_keys(gpuqo_dpsub_n_parallel);
    uninit_device_vector_joinrel gpu_scratchpad_vals(gpuqo_dpsub_n_parallel);
    uninit_device_vector_relid gpu_reduced_keys(gpuqo_dpsub_n_parallel);
    uninit_device_vector_joinrel gpu_reduced_vals(gpuqo_dpsub_n_parallel);

    STOP_TIMING(init);

#ifdef GPUQO_DEBUG
    printVector(gpu_binoms.begin(), gpu_binoms.end());    
#endif

    START_TIMING(execute);
    try{ // catch any exception in thrust
        DECLARE_TIMING(iter_init);
        DECLARE_NV_TIMING(unrank_compute);
        DECLARE_NV_TIMING(prune);
        DECLARE_NV_TIMING(scatter);
        DECLARE_NV_TIMING(build_qt);

        // iterate over the size of the resulting joinrel
        for(int i=2; i<=n_rels; i++){
            // give possibility to user to interrupt
            CHECK_FOR_INTERRUPTS();

            START_TIMING(iter_init);
            
            // calculate number of combinations of relations that make up 
            // a joinrel of size i
            uint64_t n_sets = BINOM(binoms, n_rels, n_rels, i);
            uint64_t n_joins_per_set = (1<<i) - 2;
            uint64_t tot = n_sets * n_joins_per_set;
            
            uint64_t n_joins_per_thread;
            uint64_t n_sets_per_iteration;
            uint64_t factor = gpuqo_dpsub_n_parallel / n_sets;
            if (factor < 1){ // n_sets > gpuqo_dpsub_n_parallel
                n_joins_per_thread = n_joins_per_set;
                n_sets_per_iteration = gpuqo_dpsub_n_parallel;
            } else{
                n_sets_per_iteration = n_sets;
                n_joins_per_thread = ceil_div(n_joins_per_set, factor);
            }
            
            STOP_TIMING(iter_init);

#if defined(GPUQO_DEBUG) || defined(GPUQO_PROFILE)
            printf("\nStarting iteration %d: %llu combinations\n", i, tot);
#endif
            uint64_t id_offset = 0;
            uint64_t offset = 0;
            int n_iters = 0;
            while (offset < tot){
                uint64_t n_threads;
                uint64_t threads_per_set = ceil_div(n_joins_per_set, n_joins_per_thread);
                uint64_t n_remaining_sets = (tot-offset)/n_joins_per_set;
                if (n_remaining_sets >= n_sets_per_iteration){
                    n_threads = n_sets_per_iteration*threads_per_set;
                } else {
                    n_threads = n_remaining_sets*threads_per_set;
                }   

                START_TIMING(unrank_compute);
                // fill scratchpad
                thrust::tabulate(
                    thrust::make_zip_iterator(thrust::make_tuple(
                        gpu_scratchpad_keys.begin(),
                        gpu_scratchpad_vals.begin()
                    )),
                    thrust::make_zip_iterator(thrust::make_tuple(
                        gpu_scratchpad_keys.begin()+n_threads,
                        gpu_scratchpad_vals.begin()+n_threads
                    )),
                    unrankEvaluateDPSub(
                        gpu_memo_vals.data(),
                        gpu_base_rels.data(),
                        n_rels,
                        gpu_edge_table.data(),
                        gpu_binoms.data(),
                        i,
                        id_offset,
                        n_joins_per_thread
                    ) 
                );
                STOP_TIMING(unrank_compute);

#ifdef GPUQO_DEBUG
                printf("After tabulate\n");
                printVector(gpu_scratchpad_keys.begin(), gpu_scratchpad_keys.begin()+n_threads);
                printVector(gpu_scratchpad_vals.begin(), gpu_scratchpad_vals.begin()+n_threads);
#endif

                // give possibility to user to interrupt
                CHECK_FOR_INTERRUPTS();

                thrust::pair<uninit_device_vector_relid::iterator, uninit_device_vector_joinrel::iterator> scatter_from_iters;
                thrust::pair<uninit_device_vector_relid::iterator, uninit_device_vector_joinrel::iterator> scatter_to_iters;

                if (n_joins_per_thread < n_joins_per_set){
                    START_TIMING(prune);
                    scatter_from_iters = thrust::make_pair(
                        gpu_reduced_keys.begin(),
                        gpu_reduced_vals.begin()
                    );
                    // prune to intermediate memory
                    scatter_to_iters = thrust::reduce_by_key(
                        gpu_scratchpad_keys.begin(),
                        gpu_scratchpad_keys.begin() + n_threads,
                        gpu_scratchpad_vals.begin(),
                        gpu_reduced_keys.begin(),
                        gpu_reduced_vals.begin(),
                        thrust::equal_to<uint64_t>(),
                        thrust::minimum<JoinRelation>()
                    );
                    STOP_TIMING(prune);
                } else{
                    scatter_from_iters = thrust::make_pair(
                        gpu_scratchpad_keys.begin(),
                        gpu_scratchpad_vals.begin()
                    );
                    scatter_to_iters = thrust::make_pair(
                        (gpu_scratchpad_keys.begin()+n_threads),
                        (gpu_scratchpad_vals.begin()+n_threads)
                    );
                }
    
#ifdef GPUQO_DEBUG
                printf("After reduce_by_key\n");
                printVector(scatter_from_iters.first, scatter_to_iters.first);
                printVector(scatter_from_iters.second, scatter_to_iters.second);
#endif
    
                START_TIMING(scatter);
                thrust::scatter(
                    scatter_from_iters.second,
                    scatter_to_iters.second,
                    scatter_from_iters.first,
                    gpu_memo_vals.begin()
                );
                STOP_TIMING(scatter);

                n_iters++;
                id_offset += n_threads;
                offset += n_sets_per_iteration*n_joins_per_set;
            } // loop: while(offset<tot)

#ifdef GPUQO_DEBUG
            printf("It took %d iterations\n", n_iters);
#endif
            
            PRINT_CHECKPOINT_TIMING(iter_init);
            PRINT_CHECKPOINT_TIMING(unrank_compute);
            PRINT_CHECKPOINT_TIMING(prune);
            PRINT_CHECKPOINT_TIMING(scatter);
        } // dpsub loop: for i = 2..n_rels

        START_TIMING(build_qt);
            
        buildQueryTree(out_relid, gpu_memo_vals, &out);
    
        STOP_TIMING(build_qt);
    
        PRINT_TOTAL_TIMING(iter_init);
        PRINT_TOTAL_TIMING(unrank_compute);
        PRINT_TOTAL_TIMING(prune);
        PRINT_TOTAL_TIMING(scatter);
    } catch(thrust::system_error err){
        printf("Thrust %d: %s", err.code().value(), err.what());
    }

    STOP_TIMING(execute);
    STOP_TIMING(gpuqo_dpsub);

    PRINT_TIMING(gpuqo_dpsub);
    PRINT_TIMING(init);
    PRINT_TIMING(execute);

    return out;
}
