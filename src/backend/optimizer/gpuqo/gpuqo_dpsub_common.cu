#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
 *
 * gpuqo_dpsub.cu
 *
 * src/backend/optimizer/gpuqo/gpuqo_dpsub.cu
 *
 *-------------------------------------------------------------------------
 */

#include <iostream>
#include <cmath>
#include <cstdint>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/tabulate.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/tuple.h>
#include <thrust/system/system_error.h>
#include <thrust/distance.h>

#include "optimizer/gpuqo_common.h"

#include "gpuqo.cuh"
#include "gpuqo_timing.cuh"
#include "gpuqo_debug.cuh"
#include "gpuqo_cost.cuh"
#include "gpuqo_filter.cuh"
#include "gpuqo_binomial.cuh"
#include "gpuqo_query_tree.cuh"
#include "gpuqo_dpsub.cuh"

// relsize depends on algorithm
#define RELSIZE (sizeof(JoinRelation))

PROTOTYPE_TIMING(unrank);
PROTOTYPE_TIMING(filter);
PROTOTYPE_TIMING(compute);
PROTOTYPE_TIMING(prune);
PROTOTYPE_TIMING(scatter);
PROTOTYPE_TIMING(iteration);

// User-configured option
int gpuqo_n_parallel;

template<bool CHECK_LEFT>
__device__
void try_join(JoinRelation &jr_out, RelationID l, RelationID r, 
                bool additional_predicate, join_stack_t &stack, 
                HashTable32bit &memo, GpuqoPlannerInfo* info)
{
    LOG_DEBUG("[%d, %d] try_join(%u, %u, %s)\n", 
                blockIdx.x, threadIdx.x, l, r,
                additional_predicate ? "true" : "false");

    RelationID jr = BMS32_UNION(l, r);

    bool p = additional_predicate && check_join<CHECK_LEFT>(l, r, info);

    Assert(__activemask() == WARP_MASK);

    unsigned pthBlt = __ballot_sync(WARP_MASK, !p);
    int reducedNTaken = __popc(pthBlt);
    if (LANE_ID == 0){
        LOG_DEBUG("[%d] pthBlt=%u, reducedNTaken=%d, stackTop=%d\n", W_OFFSET, pthBlt, reducedNTaken, stack.stackTop);
    }
    if (stack.stackTop >= reducedNTaken){
        int wScan = __popc(pthBlt & LANE_MASK_LE);
        int pos = W_OFFSET + stack.stackTop - wScan;
        if (!p){
            l = stack.ctxStack[pos];
            r = BMS32_DIFFERENCE(jr, l);
            LOG_DEBUG("[%d: %d] Consuming stack (%d): l=%u, r=%u\n", 
                W_OFFSET, LANE_ID, pos, l, r
            );
        } else {
            LOG_DEBUG("[%d: %d] Using local values: l=%u, r=%u\n", 
                W_OFFSET, LANE_ID, l, r
            );
        }
        stack.stackTop -= reducedNTaken;

        Assert(l != BMS32_EMPTY && r != BMS32_EMPTY);

        JoinRelation *left_rel = memo.lookup(l);
        JoinRelation *right_rel = memo.lookup(r);
        do_join(jr_out, *left_rel, *right_rel, info);

    } else{
        int wScan = __popc(~pthBlt & LANE_MASK_LE);
        int pos = W_OFFSET + stack.stackTop + wScan - 1;
        if (p){
            LOG_DEBUG("[%d: %d] Accumulating stack (%d): l=%u, r=%u\n", W_OFFSET, LANE_ID, pos, l, r);
            stack.ctxStack[pos] = l;
        }
        stack.stackTop += WARP_SIZE - reducedNTaken;
    }
    if (LANE_ID == 0){
        LOG_DEBUG("[%d] new stackTop=%d\n", W_OFFSET, stack.stackTop);
    }
}
template __device__ void try_join<true>(JoinRelation &jr_out, RelationID l, RelationID r, bool additional_predicate, join_stack_t &stack,  HashTable32bit &memo, GpuqoPlannerInfo* info);
template __device__ void try_join<false>(JoinRelation &jr_out, RelationID l, RelationID r, bool additional_predicate, join_stack_t &stack,  HashTable32bit &memo, GpuqoPlannerInfo* info);

void dpsub_prune_scatter(int threads_per_set, int n_threads, dpsub_iter_param_t &params){
    // give possibility to user to interrupt
    CHECK_FOR_INTERRUPTS();

    scatter_iter_t scatter_from_iters;
    scatter_iter_t scatter_to_iters;

    if (threads_per_set != 1){
        START_TIMING(prune);
        scatter_from_iters = thrust::make_pair(
            params.gpu_reduced_keys.begin(),
            params.gpu_reduced_vals.begin()
        );
        // prune to intermediate memory
        scatter_to_iters = thrust::reduce_by_key(
            params.gpu_scratchpad_keys.begin(),
            params.gpu_scratchpad_keys.begin() + n_threads,
            params.gpu_scratchpad_vals.begin(),
            params.gpu_reduced_keys.begin(),
            params.gpu_reduced_vals.begin(),
            thrust::equal_to<RelationID>(),
            thrust::minimum<JoinRelation>()
        );
        STOP_TIMING(prune);
    } else{
        scatter_from_iters = thrust::make_pair(
            params.gpu_scratchpad_keys.begin(),
            params.gpu_scratchpad_vals.begin()
        );
        scatter_to_iters = thrust::make_pair(
            (params.gpu_scratchpad_keys.begin()+n_threads),
            (params.gpu_scratchpad_vals.begin()+n_threads)
        );
    }

    LOG_DEBUG("After reduce_by_key\n");
    DUMP_VECTOR(scatter_from_iters.first, scatter_to_iters.first);
    DUMP_VECTOR(scatter_from_iters.second, scatter_to_iters.second);

    START_TIMING(scatter);
    params.memo->insert(
        scatter_from_iters.first.base().get(),
        scatter_from_iters.second.base().get(),
        thrust::distance(
            scatter_from_iters.first,
            scatter_to_iters.first
        )
    );
    STOP_TIMING(scatter);
}

/* gpuqo_dpsub
 *
 *	 GPU query optimization using the DP size variant.
 */
extern "C"
QueryTree*
gpuqo_dpsub(GpuqoPlannerInfo* info)
{
    DECLARE_TIMING(gpuqo_dpsub);
    DECLARE_NV_TIMING(init);
    DECLARE_NV_TIMING(execute);
    
    START_TIMING(gpuqo_dpsub);
    START_TIMING(init);

    size_t min_memo_cap = (size_t) gpuqo_min_memo_size_mb * MB / RELSIZE;
    size_t max_memo_cap = (size_t) gpuqo_max_memo_size_mb * MB / RELSIZE;
    size_t req_memo_size = 1ULL<<(info->n_rels);

    size_t memo_cap = std::min(req_memo_size*2, min_memo_cap);

    dpsub_iter_param_t params;
    params.info = info;
    params.memo = new HashTable32bit(memo_cap, max_memo_cap);
    thrust::host_vector<RelationID> ini_memo_keys(info->n_rels+1);
    thrust::host_vector<JoinRelation> ini_memo_vals(info->n_rels+1);
    thrust::device_vector<RelationID> ini_memo_keys_gpu(info->n_rels+1);
    thrust::device_vector<JoinRelation> ini_memo_vals_gpu(info->n_rels+1);

    QueryTree* out = NULL;
    params.out_relid = BMS32_EMPTY;

    for(int i=0; i<info->n_rels; i++){
        JoinRelation t;
        t.id = info->base_rels[i].id;
        t.left_relation_idx = 0; 
        t.left_relation_id = 0; 
        t.right_relation_idx = 0; 
        t.right_relation_id = 0; 
        t.cost = baserel_cost(info->base_rels[i]); 
        t.rows = info->base_rels[i].rows; 
        t.edges = info->edge_table[i];
        ini_memo_keys[i] = info->base_rels[i].id;
        ini_memo_vals[i] = t;

        params.out_relid = BMS32_UNION(params.out_relid, info->base_rels[i].id);
    }
    
    // add dummy relation
    JoinRelation dummy_jr;
    dummy_jr.id = BMS32_EMPTY;
	dummy_jr.edges = BMS32_EMPTY;
	dummy_jr.left_relation_id = BMS32_EMPTY;
	dummy_jr.right_relation_id = BMS32_EMPTY;
    dummy_jr.left_relation_ptr = NULL;
    dummy_jr.right_relation_ptr = NULL;
    dummy_jr.rows = 0.0;
	dummy_jr.cost = 0.0;
    
    ini_memo_keys[info->n_rels] = 0;
    ini_memo_vals[info->n_rels] = dummy_jr;

    // transfer base relations to GPU
    ini_memo_keys_gpu = ini_memo_keys;
    ini_memo_vals_gpu = ini_memo_vals;

    params.memo->insert(
        thrust::raw_pointer_cast(ini_memo_keys_gpu.data()), 
        thrust::raw_pointer_cast(ini_memo_vals_gpu.data()),
        info->n_rels+1
    );

    int binoms_size = (info->n_rels+1)*(info->n_rels+1);
    params.binoms = thrust::host_vector<uint32_t>(binoms_size);
    precompute_binoms(params.binoms, info->n_rels);
    params.gpu_binoms = params.binoms;

    params.scratchpad_size = (
        (
            gpuqo_scratchpad_size_mb * MB
        ) / (
            sizeof(RelationID)*gpuqo_dpsub_filter_keys_overprovisioning + 
            (sizeof(RelationID) + sizeof(JoinRelation))
        )
    );  

    if (params.scratchpad_size < gpuqo_n_parallel)
        params.scratchpad_size = gpuqo_n_parallel;

    LOG_PROFILE("Using a scratchpad of size %u\n", params.scratchpad_size);

    params.gpu_pending_keys = uninit_device_vector_relid(PENDING_KEYS_SIZE(params));
    params.gpu_scratchpad_keys = uninit_device_vector_relid(params.scratchpad_size);
    params.gpu_scratchpad_vals = uninit_device_vector_joinrel(params.scratchpad_size);
    params.gpu_reduced_keys = uninit_device_vector_relid(params.scratchpad_size);
    params.gpu_reduced_vals = uninit_device_vector_joinrel(params.scratchpad_size);

    STOP_TIMING(init);

    DUMP_VECTOR(params.gpu_binoms.begin(), params.gpu_binoms.end());    

    START_TIMING(execute);
    try{ // catch any exception in thrust
        INIT_NV_TIMING(unrank);
        INIT_NV_TIMING(filter);
        INIT_NV_TIMING(compute);
        INIT_NV_TIMING(prune);
        INIT_NV_TIMING(scatter);
        INIT_NV_TIMING(iteration);
        DECLARE_NV_TIMING(build_qt);

        // iterate over the size of the resulting joinrel
        for(int i=2; i<=info->n_rels; i++){
            // give possibility to user to interrupt
            CHECK_FOR_INTERRUPTS();
            
            // calculate number of combinations of relations that make up 
            // a joinrel of size i
            params.n_sets = BINOM(params.binoms, info->n_rels, info->n_rels, i);
            params.n_joins_per_set = ((1U)<<i);
            params.tot = ((uint64_t)params.n_sets) * params.n_joins_per_set;

            // used only if profiling is enabled
            uint32_t n_iters __attribute__((unused));
            uint64_t filter_threshold = ((uint64_t)gpuqo_n_parallel) * gpuqo_dpsub_filter_threshold;
            uint64_t csg_threshold = ((uint64_t)gpuqo_n_parallel) * gpuqo_dpsub_csg_threshold;

            START_TIMING(iteration);
            if ((gpuqo_dpsub_filter_enable && params.tot > filter_threshold) 
                    || (gpuqo_dpsub_csg_enable && params.tot > csg_threshold)){
                LOG_PROFILE("\nStarting filtered iteration %d: %llu combinations\n", i, params.tot);

                n_iters = dpsub_filtered_iteration(i, params);
            } else {
                LOG_PROFILE("\nStarting unfiltered iteration %d: %llu combinations\n", i, params.tot);

                n_iters = dpsub_unfiltered_iteration(i, params);
            }
            STOP_TIMING(iteration);

            LOG_DEBUG("It took %d iterations\n", n_iters);
            PRINT_CHECKPOINT_TIMING(unrank);
            PRINT_CHECKPOINT_TIMING(filter);
            PRINT_CHECKPOINT_TIMING(compute);
            PRINT_CHECKPOINT_TIMING(prune);
            PRINT_CHECKPOINT_TIMING(scatter);
            PRINT_TIMING(iteration);
        } // dpsub loop: for i = 2..n_rels

        START_TIMING(build_qt);
            
        buildQueryTree(params.out_relid, *params.memo, &out);
    
        STOP_TIMING(build_qt);
    
        PRINT_TOTAL_TIMING(unrank);
        PRINT_TOTAL_TIMING(filter);
        PRINT_TOTAL_TIMING(compute);
        PRINT_TOTAL_TIMING(prune);
        PRINT_TOTAL_TIMING(scatter);
    } catch(thrust::system_error err){
        printf("Thrust %d: %s\n", err.code().value(), err.what());
    }

    STOP_TIMING(execute);
    STOP_TIMING(gpuqo_dpsub);

    PRINT_TIMING(gpuqo_dpsub);
    PRINT_TIMING(init);
    PRINT_TIMING(execute);

    params.memo->free();
    delete params.memo;

    return out;
}
