#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
 *
 * gpuqo_dpsub.cu
 *
 * src/backend/optimizer/gpuqo/gpuqo_dpsub.cu
 *
 *-------------------------------------------------------------------------
 */

#include <iostream>
#include <cmath>
#include <cstdint>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/tabulate.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/tuple.h>
#include <thrust/system/system_error.h>
#include <thrust/distance.h>

#include "optimizer/gpuqo_common.h"

#include "gpuqo.cuh"
#include "gpuqo_timing.cuh"
#include "gpuqo_debug.cuh"
#include "gpuqo_cost.cuh"
#include "gpuqo_filter.cuh"
#include "gpuqo_binomial.cuh"
#include "gpuqo_query_tree.cuh"
#include "gpuqo_dpsub.cuh"

// relsize depends on algorithm
#define RELSIZE (sizeof(JoinRelation))

PROTOTYPE_TIMING(unrank);
PROTOTYPE_TIMING(filter);
PROTOTYPE_TIMING(compute);
PROTOTYPE_TIMING(prune);
PROTOTYPE_TIMING(scatter);
PROTOTYPE_TIMING(iteration);

// User-configured option
int gpuqo_n_parallel;

void dpsub_prune_scatter(int threads_per_set, int n_threads, dpsub_iter_param_t &params){
    // give possibility to user to interrupt
    CHECK_FOR_INTERRUPTS();

    scatter_iter_t scatter_from_iters;
    scatter_iter_t scatter_to_iters;

    if (threads_per_set != 1){
        START_TIMING(prune);
        scatter_from_iters = thrust::make_pair(
            params.gpu_reduced_keys.begin(),
            params.gpu_reduced_vals.begin()
        );
        // prune to intermediate memory
        scatter_to_iters = thrust::reduce_by_key(
            params.gpu_scratchpad_keys.begin(),
            params.gpu_scratchpad_keys.begin() + n_threads,
            params.gpu_scratchpad_vals.begin(),
            params.gpu_reduced_keys.begin(),
            params.gpu_reduced_vals.begin(),
            thrust::equal_to<RelationID>(),
            thrust::minimum<JoinRelation>()
        );
        STOP_TIMING(prune);
    } else{
        scatter_from_iters = thrust::make_pair(
            params.gpu_scratchpad_keys.begin(),
            params.gpu_scratchpad_vals.begin()
        );
        scatter_to_iters = thrust::make_pair(
            (params.gpu_scratchpad_keys.begin()+n_threads),
            (params.gpu_scratchpad_vals.begin()+n_threads)
        );
    }

    LOG_DEBUG("After reduce_by_key\n");
    DUMP_VECTOR(scatter_from_iters.first, scatter_to_iters.first);
    DUMP_VECTOR(scatter_from_iters.second, scatter_to_iters.second);

    dpsub_scatter(scatter_from_iters, scatter_to_iters, params);
}

void dpsub_scatter(int n_sets, dpsub_iter_param_t &params){
    // give possibility to user to interrupt
    CHECK_FOR_INTERRUPTS();

    scatter_iter_t scatter_from_iters;
    scatter_iter_t scatter_to_iters;


    scatter_from_iters = thrust::make_pair(
        params.gpu_scratchpad_keys.begin(),
        params.gpu_scratchpad_vals.begin()
    );
    scatter_to_iters = thrust::make_pair(
        (params.gpu_scratchpad_keys.begin()+n_sets),
        (params.gpu_scratchpad_vals.begin()+n_sets)
    );

    DUMP_VECTOR(scatter_from_iters.first, scatter_to_iters.first);
    DUMP_VECTOR(scatter_from_iters.second, scatter_to_iters.second);

    dpsub_scatter(scatter_from_iters, scatter_to_iters, params);
}

void dpsub_scatter(scatter_iter_t scatter_from_iters, scatter_iter_t scatter_to_iters, dpsub_iter_param_t &params){
    // give possibility to user to interrupt
    CHECK_FOR_INTERRUPTS();

    START_TIMING(scatter);
    params.memo->insert(
        scatter_from_iters.first.base().get(),
        scatter_from_iters.second.base().get(),
        thrust::distance(
            scatter_from_iters.first,
            scatter_to_iters.first
        )
    );
    STOP_TIMING(scatter);
}

/* gpuqo_dpsub
 *
 *	 GPU query optimization using the DP size variant.
 */
extern "C"
QueryTree*
gpuqo_dpsub(GpuqoPlannerInfo* info)
{
    DECLARE_TIMING(gpuqo_dpsub);
    DECLARE_NV_TIMING(init);
    DECLARE_NV_TIMING(execute);
    
    START_TIMING(gpuqo_dpsub);
    START_TIMING(init);

    size_t min_memo_cap = (size_t) gpuqo_min_memo_size_mb * MB / RELSIZE;
    size_t max_memo_cap = (size_t) gpuqo_max_memo_size_mb * MB / RELSIZE;
    size_t req_memo_size = 1ULL<<(info->n_rels);

    size_t memo_cap = std::min(req_memo_size*2, min_memo_cap);

    dpsub_iter_param_t params;
    params.info = info;
    params.memo = new HashTable32bit(memo_cap, max_memo_cap);
    thrust::host_vector<RelationID> ini_memo_keys(info->n_rels+1);
    thrust::host_vector<JoinRelation> ini_memo_vals(info->n_rels+1);
    thrust::device_vector<RelationID> ini_memo_keys_gpu(info->n_rels+1);
    thrust::device_vector<JoinRelation> ini_memo_vals_gpu(info->n_rels+1);

    QueryTree* out = NULL;
    params.out_relid = BMS32_EMPTY;

    for(int i=0; i<info->n_rels; i++){
        JoinRelation t;
        t.left_rel_id = BMS32_EMPTY; 
        t.left_rel_id = BMS32_EMPTY; 
        t.cost = baserel_cost(info->base_rels[i]); 
        t.rows = info->base_rels[i].rows; 
        ini_memo_keys[i] = info->base_rels[i].id;
        ini_memo_vals[i] = t;

        params.out_relid = BMS32_UNION(params.out_relid, info->base_rels[i].id);
    }
    
    // add dummy relation
    JoinRelation dummy_jr;
	dummy_jr.left_rel_id = BMS32_EMPTY;
	dummy_jr.right_rel_id = BMS32_EMPTY;
    dummy_jr.rows = 0.0;
	dummy_jr.cost = 0.0;
    
    ini_memo_keys[info->n_rels] = 0;
    ini_memo_vals[info->n_rels] = dummy_jr;

    // transfer base relations to GPU
    ini_memo_keys_gpu = ini_memo_keys;
    ini_memo_vals_gpu = ini_memo_vals;

    params.memo->insert(
        thrust::raw_pointer_cast(ini_memo_keys_gpu.data()), 
        thrust::raw_pointer_cast(ini_memo_vals_gpu.data()),
        info->n_rels+1
    );

    int binoms_size = (info->n_rels+1)*(info->n_rels+1);
    params.binoms = thrust::host_vector<uint32_t>(binoms_size);
    precompute_binoms(params.binoms, info->n_rels);
    params.gpu_binoms = params.binoms;

    params.scratchpad_size = (
        (
            gpuqo_scratchpad_size_mb * MB
        ) / (
            sizeof(RelationID)*gpuqo_dpsub_filter_keys_overprovisioning + 
            (sizeof(RelationID) + sizeof(JoinRelation))
        )
    );  

    if (params.scratchpad_size < gpuqo_n_parallel)
        params.scratchpad_size = gpuqo_n_parallel;

    LOG_PROFILE("Using a scratchpad of size %u\n", params.scratchpad_size);

    params.gpu_pending_keys = uninit_device_vector_relid(PENDING_KEYS_SIZE(params));
    params.gpu_scratchpad_keys = uninit_device_vector_relid(params.scratchpad_size);
    params.gpu_scratchpad_vals = uninit_device_vector_joinrel(params.scratchpad_size);
    params.gpu_reduced_keys = uninit_device_vector_relid(params.scratchpad_size);
    params.gpu_reduced_vals = uninit_device_vector_joinrel(params.scratchpad_size);

    STOP_TIMING(init);

    DUMP_VECTOR(params.gpu_binoms.begin(), params.gpu_binoms.end());    

    START_TIMING(execute);
    try{ // catch any exception in thrust
        INIT_NV_TIMING(unrank);
        INIT_NV_TIMING(filter);
        INIT_NV_TIMING(compute);
        INIT_NV_TIMING(prune);
        INIT_NV_TIMING(scatter);
        INIT_NV_TIMING(iteration);
        DECLARE_NV_TIMING(build_qt);

        // iterate over the size of the resulting joinrel
        for(int i=2; i<=info->n_rels; i++){
            // give possibility to user to interrupt
            CHECK_FOR_INTERRUPTS();
            
            // calculate number of combinations of relations that make up 
            // a joinrel of size i
            params.n_sets = BINOM(params.binoms, info->n_rels, info->n_rels, i);
            params.n_joins_per_set = ((1U)<<i);
            params.tot = ((uint64_t)params.n_sets) * params.n_joins_per_set;

            // used only if profiling is enabled
            uint32_t n_iters __attribute__((unused));
            uint64_t filter_threshold = ((uint64_t)gpuqo_n_parallel) * gpuqo_dpsub_filter_threshold;
            uint64_t csg_threshold = ((uint64_t)gpuqo_n_parallel) * gpuqo_dpsub_csg_threshold;

            START_TIMING(iteration);
            if ((gpuqo_dpsub_filter_enable && params.tot > filter_threshold) 
                    || (gpuqo_dpsub_csg_enable && params.tot > csg_threshold)){
                LOG_PROFILE("\nStarting filtered iteration %d: %llu combinations\n", i, params.tot);

                n_iters = dpsub_filtered_iteration(i, params);
            } else {
                LOG_PROFILE("\nStarting unfiltered iteration %d: %llu combinations\n", i, params.tot);

                n_iters = dpsub_unfiltered_iteration(i, params);
            }
            STOP_TIMING(iteration);

            LOG_DEBUG("It took %d iterations\n", n_iters);
            PRINT_CHECKPOINT_TIMING(unrank);
            PRINT_CHECKPOINT_TIMING(filter);
            PRINT_CHECKPOINT_TIMING(compute);
            PRINT_CHECKPOINT_TIMING(prune);
            PRINT_CHECKPOINT_TIMING(scatter);
            PRINT_TIMING(iteration);
        } // dpsub loop: for i = 2..n_rels

        START_TIMING(build_qt);
            
        dpsub_buildQueryTree(params.out_relid, *params.memo, &out);
    
        STOP_TIMING(build_qt);
    
        PRINT_TOTAL_TIMING(unrank);
        PRINT_TOTAL_TIMING(filter);
        PRINT_TOTAL_TIMING(compute);
        PRINT_TOTAL_TIMING(prune);
        PRINT_TOTAL_TIMING(scatter);
    } catch(thrust::system_error err){
        printf("Thrust %d: %s\n", err.code().value(), err.what());
    }

    STOP_TIMING(execute);
    STOP_TIMING(gpuqo_dpsub);

    PRINT_TIMING(gpuqo_dpsub);
    PRINT_TIMING(init);
    PRINT_TIMING(execute);

    params.memo->free();
    delete params.memo;

    return out;
}
