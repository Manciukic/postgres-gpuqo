#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
 *
 * gpuqo_remapper.cu
 *      implementation for class for remapping relations to other indices
 *
 * src/backend/optimizer/gpuqo/gpuqo_remapper.cu
 *
 *-------------------------------------------------------------------------
 */

#include "gpuqo_remapper.cuh"
#include "gpuqo_cost.cuh"

template<typename BitmapsetN>
Remapper<BitmapsetN>::Remapper(list<remapper_transf_el_t<BitmapsetN> > _transf) 
                                : transf(_transf) {}

template<typename BitmapsetN>
void Remapper<BitmapsetN>::countEqClasses(GpuqoPlannerInfo<BitmapsetN>* info, 
                                        int* n, int* n_sels, int *n_fk)
{
    *n = 0;
    *n_sels = 0;
    *n_fk = 0;

    for (int i = 0; i < info->n_eq_classes; i++){
        bool found = false;
        for (remapper_transf_el_t<BitmapsetN> &e : transf){
            if (info->eq_classes[i].isSubset(e.from_relid)){
                found = true;
                break;
            }
        }
        if (!found){
            (*n)++;
            (*n_fk) += info->eq_classes[i].size();
            (*n_sels) += eqClassNSels(info->eq_classes[i].size());
        }
    }
}

template<typename BitmapsetN>
BitmapsetN Remapper<BitmapsetN>::remapRelid(BitmapsetN id)
{
    BitmapsetN out = BitmapsetN(0);
    for (remapper_transf_el_t<BitmapsetN> &e : transf){
        if (e.from_relid.intersects(id)){
            out.set(e.to_idx+1);
        }
    }

    return out;
}

template<typename BitmapsetN>
BitmapsetN Remapper<BitmapsetN>::remapRelidNoComposite(BitmapsetN id)
{
    BitmapsetN out = BitmapsetN(0);
    for (remapper_transf_el_t<BitmapsetN> &e : transf){
        if (e.from_relid.size() > 1)
            continue;
        if (e.from_relid.intersects(id)){
            out.set(e.to_idx+1);
        }
    }

    return out;
}

template<typename BitmapsetN>
BitmapsetN Remapper<BitmapsetN>::remapRelidInv(BitmapsetN id)
{
    BitmapsetN out = BitmapsetN(0);
    for (remapper_transf_el_t<BitmapsetN> &e : transf){
        if (id.isSet(e.to_idx+1)){
            out |= e.from_relid;
        }
    }

    return out;
}

template<typename BitmapsetN>
void Remapper<BitmapsetN>::remapEdgeTable(BitmapsetN* edge_table_from, 
                                            BitmapsetN* edge_table_to,
                                            bool ignore_composite)
{
    for (remapper_transf_el_t<BitmapsetN> &e : transf){
        edge_table_to[e.to_idx] = BitmapsetN(0);
        
        BitmapsetN temp = e.from_relid;
        while(!temp.empty()){
            int from_idx = temp.lowestPos()-1;

            if (ignore_composite)
                edge_table_to[e.to_idx] |= remapRelidNoComposite(edge_table_from[from_idx]);
            else
                edge_table_to[e.to_idx] |= remapRelid(edge_table_from[from_idx]);

            temp.unset(from_idx+1);
        }
    }
}

template<typename BitmapsetN>
void Remapper<BitmapsetN>::remapBaseRels(
                                BaseRelation<BitmapsetN>* base_rels_from, 
                                BaseRelation<BitmapsetN>* base_rels_to)
{

    for (remapper_transf_el_t<BitmapsetN> &e : transf){
        if (e.qt != NULL){
            base_rels_to[e.to_idx].id = remapRelid(e.from_relid);
            base_rels_to[e.to_idx].rows = e.qt->rows;
            base_rels_to[e.to_idx].cost = e.qt->cost;
            base_rels_to[e.to_idx].tuples = e.qt->rows;
        } else {
            base_rels_to[e.to_idx] = base_rels_from[e.from_relid.lowestPos()-1];
            base_rels_to[e.to_idx].id = remapRelid(e.from_relid);
        }
        
        if (e.from_relid.size() == 1){
            int idx = e.from_relid.lowestPos()-1;

            base_rels_to[e.to_idx].composite = base_rels_from[idx].composite;
        } else {
            base_rels_to[e.to_idx].composite = true;
        }
    }
}

template<typename BitmapsetN>
void Remapper<BitmapsetN>::remapEqClass(BitmapsetN* eq_class_from,
                                        float* sels_from,
                                        BitmapsetN* fks_from,
                                        GpuqoPlannerInfo<BitmapsetN>* info_from,
                                        int off_sels_from, int off_fks_from,
                                        BitmapsetN* eq_class_to,
                                        float* sels_to,
                                        BitmapsetN* fks_to)
{
    *eq_class_to = remapRelid(*eq_class_from);

    int s_from = eq_class_from->size();
    int s_to = eq_class_to->size();

    for (int idx_l_to = 0; idx_l_to < s_to; idx_l_to++){
        BitmapsetN id_l_to = expandToMask(BitmapsetN::nth(idx_l_to), 
                                            *eq_class_to); 
        BitmapsetN id_l_from = remapRelidInv(id_l_to);
        int idx_l_from = (id_l_from.allLower() & *eq_class_from).size();

        for (int idx_r_to = idx_l_to+1; idx_r_to < s_to; idx_r_to++){
            BitmapsetN id_r_to = expandToMask(BitmapsetN::nth(idx_r_to), 
                                                *eq_class_to); 
            BitmapsetN id_r_from = remapRelidInv(id_r_to);
            int idx_r_from = (id_r_from.allLower() & *eq_class_from).size();

            int sels_to_idx = eqClassIndex(idx_l_to, idx_r_to, s_to);
            int sels_from_idx = eqClassIndex(idx_l_from, idx_r_from, s_from);

            if (id_l_from.size() == id_l_to.size() 
                && id_r_from.size() == id_r_to.size())
            {
                sels_to[sels_to_idx] = sels_from[sels_from_idx];
            } else {
                sels_to[sels_to_idx] = estimate_ec_selectivity(
                    *eq_class_from, off_sels_from, off_fks_from,
                    id_l_from, id_r_from, info_from
                );

            }
        }
        fks_to[idx_l_to] = remapRelidNoComposite(fks_from[idx_l_from]);
    }
}

template<typename BitmapsetN>
GpuqoPlannerInfo<BitmapsetN> *Remapper<BitmapsetN>::remapPlannerInfo(
                                        GpuqoPlannerInfo<BitmapsetN>* old_info)
{
    int n_rels = transf.size();
    int n_eq_classes, n_eq_class_sels, n_eq_class_fks; 
    countEqClasses(old_info, &n_eq_classes, &n_eq_class_sels, &n_eq_class_fks); 

    unsigned int size = sizeof(GpuqoPlannerInfo<BitmapsetN>);
	size += sizeof(BitmapsetN) * n_eq_classes;
	size += sizeof(float) * n_eq_class_sels;
	size += sizeof(BitmapsetN) * n_eq_class_fks;
	size += ceil_div(size, 8)*8; // ceil to 64 bits multiples

	char* p = new char[size];

	GpuqoPlannerInfo<BitmapsetN> *info = (GpuqoPlannerInfo<BitmapsetN>*) p;
	p += sizeof(GpuqoPlannerInfo<BitmapsetN>);

	info->size = size;
	info->n_rels = n_rels;
	info->n_iters = old_info->n_iters;

    remapEdgeTable(old_info->edge_table, info->edge_table);
    remapEdgeTable(old_info->indexed_edge_table, info->indexed_edge_table);

    if (gpuqo_spanning_tree_enable)
        remapEdgeTable(old_info->subtrees, info->subtrees);

	remapBaseRels(old_info->base_rels, info->base_rels);

	info->n_eq_classes = n_eq_classes;
	info->n_eq_class_sels = n_eq_class_sels;

	info->eq_classes = (BitmapsetN*) p;
	p += sizeof(BitmapsetN) * info->n_eq_classes;
	info->eq_class_sels = (float*) p;
	p += sizeof(float) * info->n_eq_class_sels;
	info->eq_class_fk = (BitmapsetN*) p;
	p += sizeof(BitmapsetN) * info->n_eq_class_fks;

    int off_sel = 0, off_fk = 0, old_off_sel = 0, old_off_fk = 0, j = 0;
	for (int i = 0; i < old_info->n_eq_classes; i++){
        bool found = false;
        for (remapper_transf_el_t<BitmapsetN> &e : transf){
            if (old_info->eq_classes[i].isSubset(e.from_relid)){
                found = true;
                break;
            }
        }
        if (!found){
            remapEqClass(
                &old_info->eq_classes[i], 
                &old_info->eq_class_sels[old_off_sel], 
                &old_info->eq_class_fk[old_off_fk], 
                old_info, old_off_sel, old_off_fk,
                &info->eq_classes[j], 
                &info->eq_class_sels[off_sel],
                &info->eq_class_fk[off_fk]
            );
            off_fk += info->eq_classes[j].size();
            off_sel += eqClassNSels(info->eq_classes[j].size());
            j++;
        }

        old_off_fk += old_info->eq_classes[i].size();
        old_off_sel += eqClassNSels(old_info->eq_classes[i].size());
    }

	return info;
}

template<typename BitmapsetN>
void Remapper<BitmapsetN>::remapQueryTree(QueryTree<BitmapsetN>* qt){
    if (qt->id.size() == 1){
        int idx = qt->id.lowestPos() - 1;

        for (remapper_transf_el_t<BitmapsetN> &e : transf){
            if (e.qt != NULL && e.to_idx == idx){
                *qt = *e.qt;
                
                // TODO check
                delete e.qt;

                return;
            }
        }
        // otherwise
        qt->id = remapRelidInv(qt->id);
    } else {       
        qt->id = remapRelidInv(qt->id);

        remapQueryTree(qt->left);
        remapQueryTree(qt->right);
    }
}

template class Remapper<Bitmapset32>;
template class Remapper<Bitmapset64>;
