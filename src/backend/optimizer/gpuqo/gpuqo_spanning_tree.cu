#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
 *
 * gpuqo_spanning_tree.c
 *	  procedure to extract minimum spanning tree of graph
 *
 * src/backend/optimizer/gpuqo/gpuqo_spanning_tree.c
 *
 *-------------------------------------------------------------------------
 */

#include <limits>
#include "gpuqo.cuh"
#include "gpuqo_cost.cuh"

bool gpuqo_spanning_tree_enable;

extern "C"
void minimumSpanningTree(GpuqoPlannerInfo *info){
    RelationID S = info->base_rels[0].id;
    RelationID out_relid = BMS32_EMPTY;

    EdgeMask out_edges[32];
    JoinRelation base_joinrels[32];

    for (int i=0; i < info->n_rels; i++){
        out_relid = BMS32_UNION(out_relid, info->base_rels[i].id);
        out_edges[i] = BMS32_EMPTY;

        JoinRelation t;
        t.id = info->base_rels[i].id;
        t.left_relation_idx = 0; 
        t.left_relation_id = 0; 
        t.right_relation_idx = 0; 
        t.right_relation_id = 0; 
        t.cost = baserel_cost(info->base_rels[i]); 
        t.rows = info->base_rels[i].rows; 
        t.edges = info->edge_table[i];
        base_joinrels[i] = t;
    }

    while (S != out_relid){
        float min = std::numeric_limits<float>::max();
        int argmin_in, argmin_out;
        for (int i=0; i < info->n_rels; i++){
            if (BMS32_INTERSECTS(S, BMS32_NTH(i+1))){
                RelationID edges = BMS32_DIFFERENCE(
                    info->edge_table[i],
                    S
                );
                for (int j=0; j < info->n_rels; j++){
                    if (BMS32_INTERSECTS(edges, BMS32_NTH(j+1))){
                        float sel = estimate_join_rows(
                            base_joinrels[i],
                            base_joinrels[j],
                            info
                        );
                        if (sel < min){
                            min = sel;
                            argmin_in = i;
                            argmin_out = j;
                        }
                    }
                }
            }
        }

        S = BMS32_SET(S, argmin_out+1);
        out_edges[argmin_in] = BMS32_SET(out_edges[argmin_in], argmin_out+1);
        out_edges[argmin_out] = BMS32_SET(out_edges[argmin_out], argmin_in+1);
    }
    memcpy(info->edge_table, out_edges, info->n_rels * sizeof(EdgeMask));
}

