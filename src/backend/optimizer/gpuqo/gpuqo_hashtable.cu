#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
 *
 * gpuqo_hashtable.cu
 *      implementations for GPU Hash Table
 * 
 * Derived from: https://github.com/nosferalatu/SimpleGPUHashTable
 *
 * src/backend/optimizer/gpuqo/gpuqo_hashtable.cu
 *
 *-------------------------------------------------------------------------
 */

#include <vector>

#include "gpuqo_hashtable.cuh"
#include "gpuqo_debug.cuh"

// HELPER FUNCTIONS

size_t ceilPow2(size_t x){
    size_t y = 1;
    while (y < x){
        y *= 2;
    }
    return y;
}

size_t floorPow2(size_t x){
    size_t y = 1;
    while (y*2 <= x){
        y *= 2;
    }
    return y;
}

// DEVICE FUNCTIONS IMPLEMENTATION

template <typename K, typename V, typename Kint>
__device__
V* HashTable<K,V,Kint>::lookup(K key){
    Kint slot = hash(key);
    Kint first_slot = slot;
    do {
        if (keys[slot] == key){
            LOG_DEBUG("%u: found %u (%u)\n", key, slot, hash(key));
            return &values[slot];
        } else if (keys[slot] == EMPTY){
            // NB: elements cannot be deleted!
            LOG_DEBUG("%u: not found %u (%u)\n", key, slot, hash(key));
            return NULL;
        }

        LOG_DEBUG("%u: inc %u (%u)\n", key, slot, hash(key));

        slot = (slot + 1) & (capacity-1);
    } while (slot != first_slot);

    // I checked all available positions
    return NULL;
}

template <typename K, typename V, typename Kint>
__device__
void HashTable<K,V,Kint>::insert(K key, V value){
    Kint slot = hash(key);
    Kint first_slot = slot;
    do {
        K prev = atomicCAS(&keys[slot], EMPTY, key);
        if (prev == EMPTY || prev == key){
            LOG_DEBUG("%u: found %u (%u)\n", key, slot, hash(key));
            values[slot] = value;
            return;
        }

        LOG_DEBUG("%u: inc %u (%u)\n", key, slot, hash(key));

        slot = (slot + 1) & (capacity-1);
    } while (slot != first_slot);

    // I checked all available positions
    // table is full
    assert(false);
}

// KERNELS IMPLEMENTATION

template<typename K, typename V, typename Kint>
__global__ 
void HashTable_insert(HashTable<K,V,Kint> hashtable, K* in_keys, V* in_values, size_t n)
{
    unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadid < n){
        K key = in_keys[threadid];
        V value = in_values[threadid];
        if (key != HashTable<K,V,Kint>::EMPTY){
            hashtable.insert(key, value);
            LOG_DEBUG("%u: inserted %u\n", threadid, key);
        }
    }
}

template<typename K, typename V, typename Kint>
__global__
void HashTable_lookup(HashTable<K,V,Kint> hashtable, K* in_keys, V* out_values, size_t n)
{
    unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadid < n)
    {
        K key = in_keys[threadid];
        V* val_p = hashtable.lookup(key);
        if (val_p)
            out_values[threadid] = *val_p;

        LOG_DEBUG("%u: looked up %u\n", threadid, key);  
    }
}

// HOST FUNCTIONS IMPLEMENTATION

template <typename K, typename V, typename Kint>
__host__
HashTable<K,V,Kint>::HashTable(size_t _initial_capacity, size_t _max_capacity){
    // capacity must be a multiple of 2
    capacity = ceilPow2(_initial_capacity);
    max_capacity = floorPow2(_max_capacity);
    n_elems_ub = 0;

    deviceMalloc();

    debugDump();
}

template <typename K, typename V, typename Kint>
__host__
void HashTable<K,V,Kint>::lookup(K* in_keys, V* out_values, size_t n){

    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, HashTable_lookup<K,V,Kint>, 0, 0);

    // Lookup all the keys on the hash table
    int gridsize = (n + threadblocksize - 1) / threadblocksize;
    HashTable_lookup<K,V,Kint><<<gridsize, threadblocksize>>>(*this, in_keys, out_values, n);
    
    deviceErrorCheck();
}

template <typename K, typename V, typename Kint>
__host__
void HashTable<K,V,Kint>::insert(K* in_keys, V* in_values, size_t n){
    LOG_DEBUG("HashTable::insert(%llx, %llx, %u)\n", in_keys, in_values, n);

    debugDump();

    // check if I need to grow the hashtable
    n_elems_ub += n;
    if (n_elems_ub > capacity/2 && capacity < max_capacity){
        resize(min(ceilPow2(n_elems_ub)*2, max_capacity));
    }
    
    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, HashTable_insert<K,V,Kint>, 0, 0);

    // Insert all the keys into the hash table
    int gridsize = (n + threadblocksize - 1) / threadblocksize;
    HashTable_insert<K,V,Kint><<<gridsize, threadblocksize>>>(*this, in_keys, in_values, n);
    
    deviceErrorCheck();
}


template <typename K, typename V, typename Kint>
__host__
void HashTable<K,V,Kint>::resize(size_t _capacity){
    LOG_PROFILE("resize(%u)\n", _capacity);
    size_t old_capacity = capacity;
    K* old_keys = keys;
    V* old_values = values;

    capacity = ceilPow2(_capacity);

    deviceMalloc();

    insert(old_keys, old_values, old_capacity);

    hipFree(old_keys);
    hipFree(old_values);

    deviceErrorCheck();
}

template <typename K, typename V, typename Kint>
__host__
V HashTable<K,V,Kint>::get(K key){
    V val;

    K* dev_key;
    hipMalloc(&dev_key, sizeof(K));
    hipMemcpy(dev_key, &key, sizeof(K), hipMemcpyHostToDevice);

    V* dev_val;
    hipMalloc(&dev_val, sizeof(V));

    deviceErrorCheck();

    lookup(dev_key, dev_val, 1);

    hipMemcpy(&val, dev_val, sizeof(V), hipMemcpyDeviceToHost);

    deviceErrorCheck();

    return val;
}

template <typename K, typename V, typename Kint>
__host__
void HashTable<K,V,Kint>::deviceMalloc(){
    hipMalloc(&keys, sizeof(K) * capacity);    
    hipMemset(keys, 0xff, sizeof(K) * capacity);
    LOG_DEBUG("hipMalloc(%llx, %u)\n", keys, sizeof(K) * capacity);
    
    hipMalloc(&values, sizeof(V) * capacity);    
    LOG_DEBUG("hipMalloc(%llx, %u)\n", values, sizeof(V) * capacity);

    deviceErrorCheck();
}

template <typename K, typename V, typename Kint>
__host__
void HashTable<K,V,Kint>::deviceErrorCheck(){
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err){
        printf("CUDA ERROR! %s: %s\n", 
            hipGetErrorName(err),
            hipGetErrorString(err)
        );
    }
}

template <typename K, typename V, typename Kint>
__host__
void HashTable<K,V,Kint>::debugDump(){
#ifdef GPUQO_DEBUG
    std::vector<K> local_keys(capacity);
    hipMemcpy(&local_keys[0], keys, sizeof(K)*capacity, hipMemcpyDeviceToHost);
    LOG_DEBUG("hashtable dump:\n");
    DUMP_VECTOR(local_keys.begin(), local_keys.end());
#endif
}

template <typename K, typename V, typename Kint>
__host__
void HashTable<K,V,Kint>::free(){
    hipFree(keys);    
    hipFree(values);    
}

template<>
__device__
unsigned int HashTable<uint32_t, JoinRelation, unsigned int>::hash(uint32_t k){
    k ^= k >> 16;
    k *= 0x85ebca6b;
    k ^= k >> 13;
    k *= 0xc2b2ae35;
    k ^= k >> 16;
    return k & (capacity-1);
}




// explicit specification
template class HashTable<RelationID,JoinRelation,unsigned int>;
