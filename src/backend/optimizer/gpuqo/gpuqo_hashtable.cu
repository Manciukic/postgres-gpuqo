#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
 *
 * gpuqo_hashtable.cu
 *      implementations for GPU Hash Table
 * 
 * Derived from: https://github.com/nosferalatu/SimpleGPUHashTable
 *
 * src/backend/optimizer/gpuqo/gpuqo_hashtable.cu
 *
 *-------------------------------------------------------------------------
 */

#include <vector>

#include "gpuqo_hashtable.cuh"
#include "gpuqo_debug.cuh"

// DEVICE FUNCTIONS IMPLEMENTATION

template <typename K, typename V, typename Kint>
__device__
V* HashTable<K,V,Kint>::lookup(K key){
    Kint slot = hash(key);
    while (true){
        if (keys[slot] == key){
            LOG_DEBUG("%u: found %u (%u)\n", key, slot, hash(key));
            return &values[slot];
        } else if (keys[slot] == EMPTY){
            LOG_DEBUG("%u: not found %u (%u)\n", key, slot, hash(key));
            return NULL;
        }

        LOG_DEBUG("%u: inc %u (%u)\n", key, slot, hash(key));

        slot = (slot + 1) & (capacity-1);
    }
}

template <typename K, typename V, typename Kint>
__device__
void HashTable<K,V,Kint>::insert(K key, V value){
    Kint slot = hash(key);
    while (true){
        K prev = atomicCAS(&keys[slot], EMPTY, key);
        if (prev == EMPTY || prev == key){
            LOG_DEBUG("%u: found %u (%u)\n", key, slot, hash(key));
            values[slot] = value;
            return;
        }

        LOG_DEBUG("%u: inc %u (%u)\n", key, slot, hash(key));

        slot = (slot + 1) & (capacity-1);
    }
}

// KERNELS IMPLEMENTATION

template<typename K, typename V, typename Kint>
__global__ 
void HashTable_insert(HashTable<K,V,Kint> hashtable, K* in_keys, V* in_values, size_t n)
{
    unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadid < n)
    {
        K key = in_keys[threadid];
        V value = in_values[threadid];
        hashtable.insert(key, value);
        LOG_DEBUG("%u: inserted %u\n", threadid, key);
    }
}

template<typename K, typename V, typename Kint>
__global__
void HashTable_lookup(HashTable<K,V,Kint> hashtable, K* in_keys, V* out_values, size_t n)
{
    unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadid < n)
    {
        K key = in_keys[threadid];
        V* val_p = hashtable.lookup(key);
        if (val_p)
            out_values[threadid] = *val_p;

        LOG_DEBUG("%u: looked up %u\n", threadid, key);  
    }
}

// HOST FUNCTIONS IMPLEMENTATION

template <typename K, typename V, typename Kint>
__host__
HashTable<K,V,Kint>::HashTable(size_t _capacity){
    // capacity must be a multiple of 2
    capacity = 1;
    while (capacity < _capacity)
        capacity *= 2;

        
    hipMalloc(&keys, sizeof(K) * capacity);    
    hipMemset(keys, 0xff, sizeof(K) * capacity);
    LOG_DEBUG("hipMalloc(%llx, %u)\n", keys, sizeof(K) * capacity);
    
    hipMalloc(&values, sizeof(V) * capacity);    
    LOG_DEBUG("hipMalloc(%llx, %u)\n", values, sizeof(V) * capacity);

    hipDeviceSynchronize();
#ifdef GPUQO_DEBUG
    std::vector<K> local_keys(capacity);
    hipMemcpy(&local_keys[0], keys, sizeof(K)*capacity, hipMemcpyDeviceToHost);
    LOG_DEBUG("hashtable before insert:\n");
    DUMP_VECTOR(local_keys.begin(), local_keys.end());
#endif
}

template <typename K, typename V, typename Kint>
__host__
void HashTable<K,V,Kint>::lookup(K* in_keys, V* out_values, size_t n){

    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, HashTable_lookup<K,V,Kint>, 0, 0);

    // Lookup all the keys on the hash table
    int gridsize = (n + threadblocksize - 1) / threadblocksize;
    HashTable_lookup<K,V,Kint><<<gridsize, threadblocksize>>>(*this, in_keys, out_values, n);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err){
        printf("CUDA ERROR! %s: %s\n", 
            hipGetErrorName(err),
            hipGetErrorString(err)
        );
    }
}

template <typename K, typename V, typename Kint>
__host__
void HashTable<K,V,Kint>::insert(K* in_keys, V* in_values, size_t n){
    LOG_DEBUG("HashTable::insert(%llx, %llx, %u)\n", in_keys, in_values, n);

#ifdef GPUQO_DEBUG
    std::vector<K> local_keys(capacity);
    hipMemcpy(&local_keys[0], keys, sizeof(K)*capacity, hipMemcpyDeviceToHost);
    LOG_DEBUG("hashtable before insert:\n");
    DUMP_VECTOR(local_keys.begin(), local_keys.end());
#endif
    
    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, HashTable_insert<K,V,Kint>, 0, 0);

    // Insert all the keys into the hash table
    int gridsize = (n + threadblocksize - 1) / threadblocksize;
    HashTable_insert<K,V,Kint><<<gridsize, threadblocksize>>>(*this, in_keys, in_values, n);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err){
        printf("CUDA ERROR! %s: %s\n", 
            hipGetErrorName(err),
            hipGetErrorString(err)
        );
    }
}


template <typename K, typename V, typename Kint>
__host__
V HashTable<K,V,Kint>::get(K key){
    V val;

    K* dev_key;
    hipMalloc(&dev_key, sizeof(K));
    hipMemcpy(dev_key, &key, sizeof(K), hipMemcpyHostToDevice);

    V* dev_val;
    hipMalloc(&dev_val, sizeof(V));

    lookup(dev_key, dev_val, 1);

    hipMemcpy(&val, dev_val, sizeof(V), hipMemcpyDeviceToHost);

    hipError_t err = hipGetLastError();
    if (err){
        printf("CUDA ERROR! %s: %s\n", 
            hipGetErrorName(err),
            hipGetErrorString(err)
        );
    }

    return val;
}

template <typename K, typename V, typename Kint>
__host__
void HashTable<K,V,Kint>::free(){
    hipFree(keys);    
    hipFree(values);    
}

template<>
__device__
unsigned int HashTable<uint32_t, JoinRelation, unsigned int>::hash(uint32_t k){
    k ^= k >> 16;
    k *= 0x85ebca6b;
    k ^= k >> 13;
    k *= 0xc2b2ae35;
    k ^= k >> 16;
    return k & (capacity-1);
}




// explicit specification
template class HashTable<RelationID,JoinRelation,unsigned int>;
