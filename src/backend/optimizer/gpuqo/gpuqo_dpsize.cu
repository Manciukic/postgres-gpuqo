#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
 *
 * gpuqo_dpsize.c
 *
 * src/backend/optimizer/gpuqo/gpuqo_dpsize.c
 *
 *-------------------------------------------------------------------------
 */

#include <iostream>
#include <cmath>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/tabulate.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/tuple.h>
#include <thrust/system/system_error.h>
#include <thrust/distance.h>

#include "optimizer/gpuqo_common.h"

#include "optimizer/gpuqo.cuh"
#include "optimizer/gpuqo_timing.cuh"

#define printVectorOffset(from, offset, to) { \
    auto mIter = (from); \
    mIter += (offset); \
    for(int mCount=offset; mIter != (to); ++mIter, ++mCount) \
        std::cout << mCount << " : " << *mIter << std::endl; \
}

#define printVector(from, to) printVectorOffset((from), 0, (to))

struct enumerate : public thrust::unary_function< int,thrust::tuple<RelationID, JoinRelation> >
{
    thrust::device_ptr<RelationID> memo_keys;
    thrust::device_ptr<unsigned int> partition_offsets;
    thrust::device_ptr<unsigned int> partition_sizes;
    int iid;
public:
    enumerate(
        thrust::device_ptr<RelationID> _memo_keys,
        thrust::device_ptr<unsigned int> _partition_offsets,
        thrust::device_ptr<unsigned int> _partition_sizes,
        int _iid
    ) : memo_keys(_memo_keys), partition_offsets(_partition_offsets), 
    partition_sizes(_partition_sizes), iid(_iid)
    {}

    __device__
    thrust::tuple<RelationID, JoinRelation> operator()(unsigned int cid) 
    {
        int lp = 0;
        int rp = iid - 2;
        int o = partition_sizes[lp] * partition_sizes[rp];

        while (cid >= o){
            cid -= o;
            lp++;
            rp--;
            o = partition_sizes[lp] * partition_sizes[rp];
        }

        int l = cid / partition_sizes[rp];
        int r = cid % partition_sizes[rp];

        RelationID relid;
        JoinRelation jr;

        jr.left_relation_idx = partition_offsets[lp] + l;
        jr.right_relation_idx = partition_offsets[rp] + r;

        relid = memo_keys[jr.left_relation_idx] | memo_keys[jr.right_relation_idx];

        return thrust::tuple<RelationID, JoinRelation>(relid, jr);
    }
};


struct filter : public thrust::unary_function<thrust::tuple<RelationID, JoinRelation>, bool>
{
    thrust::device_ptr<RelationID> memo_keys;
    thrust::device_ptr<JoinRelation> memo_vals;
    thrust::device_ptr<BaseRelation> base_rels;
    int n_rels;
public:
    filter(
        thrust::device_ptr<RelationID> _memo_keys,
        thrust::device_ptr<JoinRelation> _memo_vals,
        thrust::device_ptr<BaseRelation> _base_rels,
        int _n_rels
    ) : memo_keys(_memo_keys), memo_vals(_memo_vals), base_rels(_base_rels),
        n_rels(_n_rels)
    {}

    __device__
    bool operator()(thrust::tuple<RelationID, JoinRelation> t) 
    {
        RelationID relid = t.get<0>();
        JoinRelation jr = t.get<1>();

        RelationID left_id = memo_keys[jr.left_relation_idx];
        RelationID right_id = memo_keys[jr.right_relation_idx];
        JoinRelation left_rel = memo_vals[jr.left_relation_idx];
        JoinRelation right_rel = memo_vals[jr.right_relation_idx];

        if (left_id & right_id) // not disjoint
            return true;

        RelationID left_edges = 0;
        for (int i = 0; i < n_rels; i++){
            int base_relid = 1<<i;
            BaseRelation base_rel = base_rels[i];
            if (left_id & base_relid){
                left_edges |= base_rel.edges;
            }
        }

        if (left_edges & right_id) // connected
            return false;
        else // not connected
            return true;
    }
};

struct cost : public thrust::unary_function<JoinRelation,JoinRelation>
{
    thrust::device_ptr<RelationID> memo_keys;
    thrust::device_ptr<JoinRelation> memo_vals;
    thrust::device_ptr<BaseRelation> base_rels;
    int n_rels;
public:
    cost(
        thrust::device_ptr<RelationID> _memo_keys,
        thrust::device_ptr<JoinRelation> _memo_vals,
        thrust::device_ptr<BaseRelation> _base_rels,
        int _n_rels
    ) : memo_keys(_memo_keys), memo_vals(_memo_vals), base_rels(_base_rels),
        n_rels(_n_rels)
    {}

    __device__
    JoinRelation operator()(JoinRelation jr) 
    {
        RelationID left_id = memo_keys[jr.left_relation_idx];
        RelationID right_id = memo_keys[jr.right_relation_idx];
        JoinRelation left_rel = memo_vals[jr.left_relation_idx];
        JoinRelation right_rel = memo_vals[jr.right_relation_idx];

        double sel = 1.0;
        
        // maybe I can first accumulate all edges as in filter?
        // maybe I can iterate more efficiently in theese bitsets
        for (int i = 0; i < n_rels; i++){
            int base_relid = 1<<i;
            if (left_id & base_relid){
                for (int j = 0; j < n_rels; j++){
                    int peer_relid = 1<<j;
                    BaseRelation base_rel_left = base_rels[i];
                    if (base_rel_left.edges & peer_relid & right_id){
                        BaseRelation base_rel_right = base_rels[j];
                        sel *= 1.0 / base_rel_right.tuples;
                    }
                }
            }
        }
        
        double rows = sel * (double) left_rel.rows * (double) right_rel.rows;
        jr.rows = rows > 1 ? round(rows) : 1;
        jr.cost = jr.rows + left_rel.cost + right_rel.cost;

        return jr;
    }
};

void buildQueryTree(int idx, 
                            thrust::device_vector<RelationID> &gpu_memo_keys,
                            thrust::device_vector<JoinRelation> &gpu_memo_vals,
                            QueryTree **qt)
{
    JoinRelation jr = gpu_memo_vals[idx];
    RelationID relid = gpu_memo_keys[idx];

    (*qt) = (QueryTree*) malloc(sizeof(QueryTree));
    (*qt)->id = relid;
    (*qt)->left = NULL;
    (*qt)->right = NULL;
    (*qt)->rows = jr.rows;
    (*qt)->cost = jr.cost;

    if (jr.left_relation_idx == 0 && jr.right_relation_idx == 0)
        return;

    buildQueryTree(jr.left_relation_idx, gpu_memo_keys, gpu_memo_vals, &((*qt)->left));
    buildQueryTree(jr.right_relation_idx, gpu_memo_keys, gpu_memo_vals, &((*qt)->right));
}

/* gpuqo_dpsize
 *
 *	 GPU query optimization using the DP size variant.
 */
extern "C"
QueryTree*
gpuqo_dpsize(BaseRelation baserels[], int N)
{
    DECLARE_TIMING(gpuqo_dpsize);
    DECLARE_TIMING(init);
    DECLARE_TIMING(execute);
    
    START_TIMING(gpuqo_dpsize);
    START_TIMING(init);
    
    thrust::device_vector<BaseRelation> gpu_baserels(baserels, baserels + N);
    thrust::device_vector<RelationID> gpu_memo_keys(std::pow(2,N));
    thrust::device_vector<JoinRelation> gpu_memo_vals(std::pow(2,N));
    thrust::host_vector<unsigned int> partition_offsets(N);
    thrust::host_vector<unsigned int> partition_sizes(N);
    thrust::device_vector<unsigned int> gpu_partition_offsets(N);
    thrust::device_vector<unsigned int> gpu_partition_sizes(N);
    QueryTree* out = NULL;

    for(int i=0; i<N; i++){
        gpu_memo_keys[i] = baserels[i].id;
        JoinRelation t;
        t.left_relation_idx = 0; 
        t.right_relation_idx = 0; 
        t.cost = 0.2*baserels[i].rows; 
        t.rows = baserels[i].rows; 
        gpu_memo_vals[i] = t;
        partition_sizes[i] = i == 0 ? N : 0;
        partition_offsets[i] = i == 1 ? N : 0;
    }
    gpu_partition_offsets = partition_offsets;
    gpu_partition_sizes = partition_sizes;

    STOP_TIMING(init);

#ifdef GPUQO_DEBUG
    printVector(gpu_memo_keys.begin(), gpu_memo_keys.begin() + N);
    printVector(gpu_memo_vals.begin(), gpu_memo_vals.begin() + N);    
#endif

    START_TIMING(execute);
    try{
        DECLARE_TIMING(iter_init);
        DECLARE_TIMING(enumerate);
        DECLARE_TIMING(filter);
        DECLARE_TIMING(sort);
        DECLARE_TIMING(compute_prune);
        DECLARE_TIMING(update_offsets);
        DECLARE_TIMING(build_qt);

        for(int i=2; i<=N; i++){
            START_TIMING(iter_init);
            // calculate size of required temp space
            int n_combinations = 0;
            for (int j=1; j<i; j++){
                n_combinations += partition_sizes[j-1] * partition_sizes[i-j-1];
            }

#ifdef GPUQO_DEBUG
            printf("Starting iteration %d: %d combinations\n", i, n_combinations);
#endif
            // allocate temp scratchpad
            thrust::device_vector<RelationID> gpu_scratchpad_keys(n_combinations);
            thrust::device_vector<JoinRelation> gpu_scratchpad_vals(n_combinations);

            STOP_TIMING(iter_init);
            START_TIMING(enumerate);
            
            // fill scratchpad
            thrust::tabulate(
                thrust::make_zip_iterator(thrust::make_tuple(
                    gpu_scratchpad_keys.begin(),
                    gpu_scratchpad_vals.begin()
                )),
                thrust::make_zip_iterator(thrust::make_tuple(
                    gpu_scratchpad_keys.end(),
                    gpu_scratchpad_vals.end()
                )),
                enumerate(
                    gpu_memo_keys.data(), 
                    gpu_partition_offsets.data(), 
                    gpu_partition_sizes.data(), 
                    i
                )
            );

            STOP_TIMING(enumerate);

#ifdef GPUQO_DEBUG
            printf("After tabulate\n");
            printVector(gpu_scratchpad_keys.begin(), gpu_scratchpad_keys.end());
            printVector(gpu_scratchpad_vals.begin(), gpu_scratchpad_vals.end());
#endif

            START_TIMING(filter);
            // filter out invalid pairs
            auto newEnd = thrust::remove_if(
                thrust::make_zip_iterator(thrust::make_tuple(
                    gpu_scratchpad_keys.begin(),
                    gpu_scratchpad_vals.begin()
                )),
                thrust::make_zip_iterator(thrust::make_tuple(
                    gpu_scratchpad_keys.end(),
                    gpu_scratchpad_vals.end()
                )),
                filter(
                    gpu_memo_keys.data(), 
                    gpu_memo_vals.data(),
                    gpu_baserels.data(), 
                    N
                )
            );

            STOP_TIMING(filter);

#ifdef GPUQO_DEBUG
            printf("After remove_if\n");
            printVector(gpu_scratchpad_keys.begin(), newEnd.get_iterator_tuple().get<0>());
            printVector(gpu_scratchpad_vals.begin(), newEnd.get_iterator_tuple().get<1>());
#endif

            START_TIMING(sort);

            // sort by key (prepare for pruning)
            thrust::sort_by_key(
                gpu_scratchpad_keys.begin(),
                newEnd.get_iterator_tuple().get<0>(),
                gpu_scratchpad_vals.begin()
            );

            STOP_TIMING(sort);

#ifdef GPUQO_DEBUG
            printf("After sort_by_key\n");
            printVector(gpu_scratchpad_keys.begin(), newEnd.get_iterator_tuple().get<0>());
            printVector(gpu_scratchpad_vals.begin(), newEnd.get_iterator_tuple().get<1>());
#endif

            START_TIMING(compute_prune);

            // calculate cost, prune and copy to table
            auto out_iters = thrust::reduce_by_key(
                gpu_scratchpad_keys.begin(),
                newEnd.get_iterator_tuple().get<0>(),
                thrust::make_transform_iterator(
                    gpu_scratchpad_vals.begin(),
                    cost(
                        gpu_memo_keys.data(), 
                        gpu_memo_vals.data(),
                        gpu_baserels.data(),
                        N
                    )
                ),
                gpu_memo_keys.begin()+partition_offsets[i-1],
                gpu_memo_vals.begin()+partition_offsets[i-1],
                thrust::equal_to<unsigned int>(),
                thrust::minimum<JoinRelation>()
            );

            STOP_TIMING(compute_prune);

#ifdef GPUQO_DEBUG
            printf("After reduce_by_key\n");
            printVector(gpu_memo_keys.begin(), out_iters.first);
            printVector(gpu_memo_vals.begin(), out_iters.second);
#endif

            START_TIMING(update_offsets);

            // update ps and po
            partition_sizes[i-1] = thrust::distance(
                gpu_memo_keys.begin()+partition_offsets[i-1],
                out_iters.first
            ); // TODO check inclusive/exclusive
            gpu_partition_sizes[i-1] = partition_sizes[i-1];
            
            if (i < N){
                partition_offsets[i] = partition_sizes[i-1] + partition_offsets[i-1];
                gpu_partition_offsets[i] = partition_offsets[i];
            }

            STOP_TIMING(update_offsets);

#ifdef GPUQO_DEBUG
            printf("After partition_*\n");
            printVector(partition_sizes.begin(), partition_sizes.end());
            printVector(partition_offsets.begin(), partition_offsets.end());
#endif

            PRINT_TIMING(iter_init);
            PRINT_TIMING(enumerate);
            PRINT_TIMING(filter);
            PRINT_TIMING(sort);
            PRINT_TIMING(compute_prune);
            PRINT_TIMING(update_offsets);
        }

        START_TIMING(build_qt);
            
        buildQueryTree(partition_offsets[N-1], gpu_memo_keys, gpu_memo_vals, &out);
    
        STOP_TIMING(build_qt);
    
        PRINT_TOTAL_TIMING(iter_init);
        PRINT_TOTAL_TIMING(enumerate);
        PRINT_TOTAL_TIMING(filter);
        PRINT_TOTAL_TIMING(sort);
        PRINT_TOTAL_TIMING(compute_prune);
        PRINT_TOTAL_TIMING(update_offsets);
        PRINT_TOTAL_TIMING(build_qt);
    } catch(thrust::system_error err){
        printf("Thrust %d: %s", err.code().value(), err.what());
    }

    STOP_TIMING(execute);
    STOP_TIMING(gpuqo_dpsize);

    PRINT_TIMING(gpuqo_dpsize);
    PRINT_TIMING(init);
    PRINT_TIMING(execute);

    return out;
}
